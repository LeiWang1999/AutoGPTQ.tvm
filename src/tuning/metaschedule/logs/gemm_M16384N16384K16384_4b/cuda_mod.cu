#include "hip/hip_runtime.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__ ((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)                              \
  TVM_XINLINE RTYPE operator OP (half a, half b) {                \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (half a, T b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (T a, half b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                                \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const T& a) {                    \
    return *this = half(float(*this) OP float(a));                \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const volatile T& a) volatile {  \
    return *this = half(float(*this) OP float(a));                \
  }

class TVM_ALIGNED(2) half {
 public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value) {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float& value) { constructor(value); }
  TVM_XINLINE explicit half(const double& value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const long long& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t& value) { constructor(value); }

  TVM_XINLINE operator float() const {                          \
    return float(half2float(half_));                            \
  }                                                             \
  TVM_XINLINE operator float() const volatile {                 \
    return float(half2float(half_));                            \
  }


  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+() {
    return *this;
  }

  TVM_XINLINE half operator-() {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half& a) {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half& a) volatile {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) volatile {
    return *this = half(a);
  }

 private:
  union Bits {
    float f;
    int32_t si;
    uint32_t ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits);   // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;   // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;   // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15;   // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;   // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;   // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;   // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000;  // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift;   // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

  static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
  static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400;  // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float& value) const {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
    const volatile float& value) const volatile {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t& value) const {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
    const volatile uint16_t& value) const volatile {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template<typename T>
  TVM_XINLINE void constructor(const T& value) {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a) {
  return half(a);
}
#endif


// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {   \
  float tmp_x = __half2float(x);                                          \
  float tmp_y = __half2float(y);                                          \
  float result = FP32_MATH_NAME(tmp_x, tmp_y);                            \
  return __float2half(result);                                            \
}

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x) {          \
  float tmp_x = __half2float(x);                                         \
  float result = FP32_MATH_NAME(tmp_x);                                  \
  return __float2half(result);                                           \
}

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

#endif
#include <mma.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) main_kernel0(half* __restrict__ A, signed char* __restrict__ B, half* __restrict__ C) {
  extern __shared__ uchar buf_dyn_shmem[];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_reindex_shared_dyn_wmma_accumulator[16];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_reindex_shared_dyn_wmma_matrix_a[8];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_decompress_reindex_shared_dyn_wmma_matrix_b[8];
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[0], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[1], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[4], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[5], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[2], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[3], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[6], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[7], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[8], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[9], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[12], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[13], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[10], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[11], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[14], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(C_reindex_shared_dyn_wmma_accumulator[15], 0.000000e+00f);
  for (int ax2_0_0 = 0; ax2_0_0 < 512; ++ax2_0_0) {
    __syncthreads();
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 5120)) = *(uint2*)(A + (((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 6400)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 524288));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 7680)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1048576));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 8960)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1572864));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 10240)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2097152));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 11520)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2621440));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 12800)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3145728));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 160) + ((((int)threadIdx.x) >> 3) * 40)) + ((((int)threadIdx.x) & 7) * 4)) + 14080)) = *(uint2*)(A + ((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3670016));
    uint1 __1;
    int2 __2;
      int2 __3;
        int2 v_ = make_int2(((int)B[(((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15))]), ((int)B[(((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15))]));
        int2 v__1 = make_int2((0)+(4*0), (0)+(4*1));
        __3.x = (v_.x >> v__1.x);
        __3.y = (v_.y >> v__1.y);
      int2 v__2 = make_int2(15, 15);
      __2.x = (__3.x & v__2.x);
      __2.y = (__3.y & v__2.y);
    ((half2*)(&(__1.x)))->x = (half)(__2.x);
    ((half2*)(&(__1.x)))->y = (half)(__2.y);
    *(uint1*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2))) = __1;
    uint1 __4;
    int2 __5;
      int2 __6;
        int2 v__3 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 131072)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 131072)]));
        int2 v__4 = make_int2((0)+(4*0), (0)+(4*1));
        __6.x = (v__3.x >> v__4.x);
        __6.y = (v__3.y >> v__4.y);
      int2 v__5 = make_int2(15, 15);
      __5.x = (__6.x & v__5.x);
      __5.y = (__6.y & v__5.y);
    ((half2*)(&(__4.x)))->x = (half)(__5.x);
    ((half2*)(&(__4.x)))->y = (half)(__5.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 640)) = __4;
    uint1 __7;
    int2 __8;
      int2 __9;
        int2 v__6 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 262144)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 262144)]));
        int2 v__7 = make_int2((0)+(4*0), (0)+(4*1));
        __9.x = (v__6.x >> v__7.x);
        __9.y = (v__6.y >> v__7.y);
      int2 v__8 = make_int2(15, 15);
      __8.x = (__9.x & v__8.x);
      __8.y = (__9.y & v__8.y);
    ((half2*)(&(__7.x)))->x = (half)(__8.x);
    ((half2*)(&(__7.x)))->y = (half)(__8.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 1280)) = __7;
    uint1 __10;
    int2 __11;
      int2 __12;
        int2 v__9 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 393216)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 393216)]));
        int2 v__10 = make_int2((0)+(4*0), (0)+(4*1));
        __12.x = (v__9.x >> v__10.x);
        __12.y = (v__9.y >> v__10.y);
      int2 v__11 = make_int2(15, 15);
      __11.x = (__12.x & v__11.x);
      __11.y = (__12.y & v__11.y);
    ((half2*)(&(__10.x)))->x = (half)(__11.x);
    ((half2*)(&(__10.x)))->y = (half)(__11.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 1920)) = __10;
    uint1 __13;
    int2 __14;
      int2 __15;
        int2 v__12 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 524288)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 524288)]));
        int2 v__13 = make_int2((0)+(4*0), (0)+(4*1));
        __15.x = (v__12.x >> v__13.x);
        __15.y = (v__12.y >> v__13.y);
      int2 v__14 = make_int2(15, 15);
      __14.x = (__15.x & v__14.x);
      __14.y = (__15.y & v__14.y);
    ((half2*)(&(__13.x)))->x = (half)(__14.x);
    ((half2*)(&(__13.x)))->y = (half)(__14.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 2560)) = __13;
    uint1 __16;
    int2 __17;
      int2 __18;
        int2 v__15 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 655360)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 655360)]));
        int2 v__16 = make_int2((0)+(4*0), (0)+(4*1));
        __18.x = (v__15.x >> v__16.x);
        __18.y = (v__15.y >> v__16.y);
      int2 v__17 = make_int2(15, 15);
      __17.x = (__18.x & v__17.x);
      __17.y = (__18.y & v__17.y);
    ((half2*)(&(__16.x)))->x = (half)(__17.x);
    ((half2*)(&(__16.x)))->y = (half)(__17.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 3200)) = __16;
    uint1 __19;
    int2 __20;
      int2 __21;
        int2 v__18 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 786432)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 786432)]));
        int2 v__19 = make_int2((0)+(4*0), (0)+(4*1));
        __21.x = (v__18.x >> v__19.x);
        __21.y = (v__18.y >> v__19.y);
      int2 v__20 = make_int2(15, 15);
      __20.x = (__21.x & v__20.x);
      __20.y = (__21.y & v__20.y);
    ((half2*)(&(__19.x)))->x = (half)(__20.x);
    ((half2*)(&(__19.x)))->y = (half)(__20.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 3840)) = __19;
    uint1 __22;
    int2 __23;
      int2 __24;
        int2 v__21 = make_int2(((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 917504)]), ((int)B[((((((((((int)blockIdx.y) & 63) * 2097152) + ((((int)blockIdx.x) & 1) * 1048576)) + (((int)threadIdx.y) * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (ax2_0_0 * 16)) + (((int)threadIdx.x) & 15)) + 917504)]));
        int2 v__22 = make_int2((0)+(4*0), (0)+(4*1));
        __24.x = (v__21.x >> v__22.x);
        __24.y = (v__21.y >> v__22.y);
      int2 v__23 = make_int2(15, 15);
      __23.x = (__24.x & v__23.x);
      __23.y = (__24.y & v__23.y);
    ((half2*)(&(__22.x)))->x = (half)(__23.x);
    ((half2*)(&(__22.x)))->y = (half)(__23.y);
    *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 80) + ((((int)threadIdx.x) >> 4) * 40)) + ((((int)threadIdx.x) & 15) * 2)) + 4480)) = __22;
    __syncthreads();
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[0], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 5120)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[1], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 5136)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[2], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 5760)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[3], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 5776)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[4], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 6400)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[5], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 6416)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[6], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 7040)])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[7], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) >> 1) * 2560) + 7056)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[0], (&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) & 1) * 2560)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[1], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 16)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[2], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 640)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[3], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 656)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[4], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 1280)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[5], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 1296)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[6], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 1920)])), 40);
    nvcuda::wmma::load_matrix_sync(B_decompress_reindex_shared_dyn_wmma_matrix_b[7], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) & 1) * 2560) + 1936)])), 40);
    for (int ax0_0_3 = 0; ax0_0_3 < 2; ++ax0_0_3) {
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[(ax0_0_3 * 8)], A_reindex_shared_dyn_wmma_matrix_a[(ax0_0_3 * 4)], B_decompress_reindex_shared_dyn_wmma_matrix_b[0], C_reindex_shared_dyn_wmma_accumulator[(ax0_0_3 * 8)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 1)], A_reindex_shared_dyn_wmma_matrix_a[(ax0_0_3 * 4)], B_decompress_reindex_shared_dyn_wmma_matrix_b[2], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 1)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 4)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 2)], B_decompress_reindex_shared_dyn_wmma_matrix_b[0], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 4)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 5)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 2)], B_decompress_reindex_shared_dyn_wmma_matrix_b[2], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 5)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[(ax0_0_3 * 8)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 1)], B_decompress_reindex_shared_dyn_wmma_matrix_b[1], C_reindex_shared_dyn_wmma_accumulator[(ax0_0_3 * 8)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 1)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 1)], B_decompress_reindex_shared_dyn_wmma_matrix_b[3], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 1)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 4)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 3)], B_decompress_reindex_shared_dyn_wmma_matrix_b[1], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 4)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 5)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 3)], B_decompress_reindex_shared_dyn_wmma_matrix_b[3], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 5)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 2)], A_reindex_shared_dyn_wmma_matrix_a[(ax0_0_3 * 4)], B_decompress_reindex_shared_dyn_wmma_matrix_b[4], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 2)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 3)], A_reindex_shared_dyn_wmma_matrix_a[(ax0_0_3 * 4)], B_decompress_reindex_shared_dyn_wmma_matrix_b[6], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 3)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 6)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 2)], B_decompress_reindex_shared_dyn_wmma_matrix_b[4], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 6)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 7)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 2)], B_decompress_reindex_shared_dyn_wmma_matrix_b[6], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 7)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 2)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 1)], B_decompress_reindex_shared_dyn_wmma_matrix_b[5], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 2)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 3)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 1)], B_decompress_reindex_shared_dyn_wmma_matrix_b[7], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 3)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 6)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 3)], B_decompress_reindex_shared_dyn_wmma_matrix_b[5], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 6)]);
      nvcuda::wmma::mma_sync(C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 7)], A_reindex_shared_dyn_wmma_matrix_a[((ax0_0_3 * 4) + 3)], B_decompress_reindex_shared_dyn_wmma_matrix_b[7], C_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 7)]);
    }
  }
  for (int ax2 = 0; ax2 < 4; ++ax2) {
    __syncthreads();
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 1024) + 5120)])), C_reindex_shared_dyn_wmma_accumulator[(ax2 * 4)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 1024) + 5376)])), C_reindex_shared_dyn_wmma_accumulator[((ax2 * 4) + 1)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 1024) + 5632)])), C_reindex_shared_dyn_wmma_accumulator[((ax2 * 4) + 2)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 1024) + 5888)])), C_reindex_shared_dyn_wmma_accumulator[((ax2 * 4) + 3)], 16, nvcuda::wmma::mem_row_major);
    __syncthreads();
    *(uint2*)(C + ((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4))) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 5120));
    *(uint2*)(C + (((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 64)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 6144));
    *(uint2*)(C + (((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 1048576)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 7168));
    *(uint2*)(C + ((((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 3072) >> 11) * 1048576)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 64)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 8192));
    *(uint2*)(C + (((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 2097152)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 9216));
    *(uint2*)(C + ((((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 5120) >> 11) * 1048576)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 64)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 10240));
    *(uint2*)(C + (((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 3145728)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 11264));
    *(uint2*)(C + ((((((((((((((int)blockIdx.y) >> 6) * 16777216) + ((((int)blockIdx.x) >> 1) * 4194304)) + (((((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 7168) >> 11) * 1048576)) + (ax2 * 262144)) + ((((int)threadIdx.y) & 1) * 131072)) + ((((int)threadIdx.x) >> 2) * 16384)) + ((((int)blockIdx.y) & 63) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 64)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 12288));
  }
}

