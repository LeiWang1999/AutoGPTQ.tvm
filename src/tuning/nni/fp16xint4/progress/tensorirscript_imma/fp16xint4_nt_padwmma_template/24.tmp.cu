#include "hip/hip_runtime.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__ ((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)                              \
  TVM_XINLINE RTYPE operator OP (half a, half b) {                \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (half a, T b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (T a, half b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                                \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const T& a) {                    \
    return *this = half(float(*this) OP float(a));                \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const volatile T& a) volatile {  \
    return *this = half(float(*this) OP float(a));                \
  }

class TVM_ALIGNED(2) half {
 public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value) {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float& value) { constructor(value); }
  TVM_XINLINE explicit half(const double& value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const long long& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t& value) { constructor(value); }

  TVM_XINLINE operator float() const {                          \
    return float(half2float(half_));                            \
  }                                                             \
  TVM_XINLINE operator float() const volatile {                 \
    return float(half2float(half_));                            \
  }


  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+() {
    return *this;
  }

  TVM_XINLINE half operator-() {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half& a) {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half& a) volatile {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) volatile {
    return *this = half(a);
  }

 private:
  union Bits {
    float f;
    int32_t si;
    uint32_t ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits);   // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;   // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;   // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15;   // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;   // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;   // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;   // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000;  // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift;   // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

  static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
  static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400;  // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float& value) const {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
    const volatile float& value) const volatile {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t& value) const {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
    const volatile uint16_t& value) const volatile {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template<typename T>
  TVM_XINLINE void constructor(const T& value) {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a) {
  return half(a);
}
#endif


// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {   \
  float tmp_x = __half2float(x);                                          \
  float tmp_y = __half2float(y);                                          \
  float result = FP32_MATH_NAME(tmp_x, tmp_y);                            \
  return __float2half(result);                                            \
}

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x) {          \
  float tmp_x = __half2float(x);                                         \
  float result = FP32_MATH_NAME(tmp_x);                                  \
  return __float2half(result);                                           \
}

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

#endif
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif
__device__ int4 make_int4(signed char _0, signed char _1, signed char _2, signed char _3, signed char _4, signed char _5, signed char _6, signed char _7, signed char _8, signed char _9, signed char _10, signed char _11, signed char _12, signed char _13, signed char _14, signed char _15) {
  return make_int4(
    *((int *)&make_char4(_0, _1, _2, _3)),
    *((int *)&make_char4(_4, _5, _6, _7)),
    *((int *)&make_char4(_8, _9, _10, _11)),
    *((int *)&make_char4(_12, _13, _14, _15)));
}
#include <mma.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#if (__CUDACC_VER_MAJOR__ >= 11) 
#define TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST 1
#else
#define TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(half* __restrict__ A, signed char* __restrict__ B, half* __restrict__ C) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[8];
  __shared__ half A_shared[2048];
  __shared__ half B_decompress_shared[16384];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[1];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_decompress_shared_wmma_matrix_b[8];
  for (int j_0_2_init = 0; j_0_2_init < 8; ++j_0_2_init) {
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[j_0_2_init], 0.000000e+00f);
  }
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((ax0_ax1_fused_2 * 256) + (((int)threadIdx.x) * 8)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((ax0_ax1_fused_2 * 256) + (((int)threadIdx.x) * 8))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.y) * 262144) + (ax0_ax1_fused_2 * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16)
    );
  }
  }
  for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 64; ++ax0_ax1_fused_2_1) {
    uint2 __1;
    int4 __2;
      int4 __3;
        int4 __4;
        int4 __5;
          int4 v_ = make_int4((((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_1 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)threadIdx.x) & 15) * 2)), (((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_1 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)threadIdx.x) & 15) * 2)), (((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_1 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)threadIdx.x) & 15) * 2)), (((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_1 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)threadIdx.x) & 15) * 2)));
          int4 __6;
            int4 v__1 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
            int4 v__2 = make_int4(2, 2, 2, 2);
            __6.x = (v__1.x%v__2.x);
            __6.y = (v__1.y%v__2.y);
            __6.z = (v__1.z%v__2.z);
            __6.w = (v__1.w%v__2.w);
          int4 __7;
            int4 v__3 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
            int4 v__4 = make_int4(2, 2, 2, 2);
            __7.x = (v__3.x/v__4.x);
            __7.y = (v__3.y/v__4.y);
            __7.z = (v__3.z/v__4.z);
            __7.w = (v__3.w/v__4.w);
          int4 __8;
          ushort4 __9;
            ushort4 __10;
              ushort4 __11;
                int4 v__5 = make_int4(2, 2, 2, 2);
                int4 v__6 = make_int4(0, 0, 0, 0);
                __11.x = (v__5.x>=v__6.x);
                __11.y = (v__5.y>=v__6.y);
                __11.z = (v__5.z>=v__6.z);
                __11.w = (v__5.w>=v__6.w);
              ushort4 __12;
                int4 v__7 = make_int4(0, 0, 0, 0);
                __12.x = (__6.x>=v__7.x);
                __12.y = (__6.y>=v__7.y);
                __12.z = (__6.z>=v__7.z);
                __12.w = (__6.w>=v__7.w);
              __10.x = (__11.x&&__12.x);
              __10.y = (__11.y&&__12.y);
              __10.z = (__11.z&&__12.z);
              __10.w = (__11.w&&__12.w);
            ushort4 __13;
              ushort4 __14;
                int4 v__8 = make_int4(2, 2, 2, 2);
                int4 v__9 = make_int4(0, 0, 0, 0);
                __14.x = (v__8.x<v__9.x);
                __14.y = (v__8.y<v__9.y);
                __14.z = (v__8.z<v__9.z);
                __14.w = (v__8.w<v__9.w);
              ushort4 __15;
                int4 v__10 = make_int4(0, 0, 0, 0);
                __15.x = (__6.x<=v__10.x);
                __15.y = (__6.y<=v__10.y);
                __15.z = (__6.z<=v__10.z);
                __15.w = (__6.w<=v__10.w);
              __13.x = (__14.x&&__15.x);
              __13.y = (__14.y&&__15.y);
              __13.z = (__14.z&&__15.z);
              __13.w = (__14.w&&__15.w);
            __9.x = (__10.x||__13.x);
            __9.y = (__10.y||__13.y);
            __9.z = (__10.z||__13.z);
            __9.w = (__10.w||__13.w);
          int4 __16;
            int4 v__11 = make_int4(1, 1, 1, 1);
            __16.x = (__7.x-v__11.x);
            __16.y = (__7.y-v__11.y);
            __16.z = (__7.z-v__11.z);
            __16.w = (__7.w-v__11.w);
          __8.x = (bool(__9.x)?__7.x:__16.x);
          __8.y = (bool(__9.y)?__7.y:__16.y);
          __8.z = (bool(__9.z)?__7.z:__16.z);
          __8.w = (bool(__9.w)?__7.w:__16.w);
          __5.x = (v_.x+__8.x);
          __5.y = (v_.y+__8.y);
          __5.z = (v_.z+__8.z);
          __5.w = (v_.w+__8.w);
        int v__12 = ((0x000000ff << 0) & (B[__5.x] << 0))|((0x000000ff << 8) & (B[__5.y] << 8))|((0x000000ff << 16) & (B[__5.z] << 16))|((0x000000ff << 24) & (B[__5.w] << 24));
        __4.x = (int)(((char)(v__12 >> 0)));
        __4.y = (int)(((char)(v__12 >> 8)));
        __4.z = (int)(((char)(v__12 >> 16)));
        __4.w = (int)(((char)(v__12 >> 24)));
        int4 __17;
          int4 __18;
            int4 v__13 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
            int4 v__14 = make_int4(2, 2, 2, 2);
            __18.x = (v__13.x%v__14.x);
            __18.y = (v__13.y%v__14.y);
            __18.z = (v__13.z%v__14.z);
            __18.w = (v__13.w%v__14.w);
          int4 __19;
          ushort4 __20;
            ushort4 __21;
              ushort4 __22;
                int4 v__15 = make_int4(2, 2, 2, 2);
                int4 v__16 = make_int4(0, 0, 0, 0);
                __22.x = (v__15.x>=v__16.x);
                __22.y = (v__15.y>=v__16.y);
                __22.z = (v__15.z>=v__16.z);
                __22.w = (v__15.w>=v__16.w);
              ushort4 __23;
                int4 v__17 = make_int4(0, 0, 0, 0);
                __23.x = (__18.x>=v__17.x);
                __23.y = (__18.y>=v__17.y);
                __23.z = (__18.z>=v__17.z);
                __23.w = (__18.w>=v__17.w);
              __21.x = (__22.x&&__23.x);
              __21.y = (__22.y&&__23.y);
              __21.z = (__22.z&&__23.z);
              __21.w = (__22.w&&__23.w);
            ushort4 __24;
              ushort4 __25;
                int4 v__18 = make_int4(2, 2, 2, 2);
                int4 v__19 = make_int4(0, 0, 0, 0);
                __25.x = (v__18.x<v__19.x);
                __25.y = (v__18.y<v__19.y);
                __25.z = (v__18.z<v__19.z);
                __25.w = (v__18.w<v__19.w);
              ushort4 __26;
                int4 v__20 = make_int4(0, 0, 0, 0);
                __26.x = (__18.x<=v__20.x);
                __26.y = (__18.y<=v__20.y);
                __26.z = (__18.z<=v__20.z);
                __26.w = (__18.w<=v__20.w);
              __24.x = (__25.x&&__26.x);
              __24.y = (__25.y&&__26.y);
              __24.z = (__25.z&&__26.z);
              __24.w = (__25.w&&__26.w);
            __20.x = (__21.x||__24.x);
            __20.y = (__21.y||__24.y);
            __20.z = (__21.z||__24.z);
            __20.w = (__21.w||__24.w);
          int4 __27;
            int4 v__21 = make_int4(2, 2, 2, 2);
            __27.x = (__18.x+v__21.x);
            __27.y = (__18.y+v__21.y);
            __27.z = (__18.z+v__21.z);
            __27.w = (__18.w+v__21.w);
          __19.x = (bool(__20.x)?__18.x:__27.x);
          __19.y = (bool(__20.y)?__18.y:__27.y);
          __19.z = (bool(__20.z)?__18.z:__27.z);
          __19.w = (bool(__20.w)?__18.w:__27.w);
          int4 v__22 = make_int4(4, 4, 4, 4);
          __17.x = (__19.x*v__22.x);
          __17.y = (__19.y*v__22.y);
          __17.z = (__19.z*v__22.z);
          __17.w = (__19.w*v__22.w);
        __3.x = (__4.x >> __17.x);
        __3.y = (__4.y >> __17.y);
        __3.z = (__4.z >> __17.z);
        __3.w = (__4.w >> __17.w);
      int4 v__23 = make_int4(15, 15, 15, 15);
      __2.x = (__3.x & v__23.x);
      __2.y = (__3.y & v__23.y);
      __2.z = (__3.z & v__23.z);
      __2.w = (__3.w & v__23.w);
    ((half2*)(&(__1.x)))->x = (half)(__2.x);
    ((half2*)(&(__1.x)))->y = (half)(__2.y);
    ((half2*)(&(__1.y)))->x = (half)(__2.z);
    ((half2*)(&(__1.y)))->y = (half)(__2.w);
    *(uint2*)(B_decompress_shared + ((ax0_ax1_fused_2_1 * 128) + (((int)threadIdx.x) * 4))) = __1;
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_0 = 0; k_0_0 < 255; ++k_0_0) {
    __syncthreads();
    for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 4; ++ax0_ax1_fused_2_2) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((k_0_0 + 1) & 1) * 1024) + (ax0_ax1_fused_2_2 * 256)) + (((int)threadIdx.x) * 8)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_0 + 1) & 1) * 1024) + (ax0_ax1_fused_2_2 * 256)) + (((int)threadIdx.x) * 8))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.y) * 262144) + (ax0_ax1_fused_2_2 * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16)
    );
  }
    }
    for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 64; ++ax0_ax1_fused_2_3) {
      uint2 __28;
      int4 __29;
        int4 __30;
          int4 __31;
          int4 __32;
            int4 __33;
              int4 v__24 = make_int4(((((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_3 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), ((((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_3 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), ((((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_3 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), ((((((((int)blockIdx.x) * 8388608) + (((int)blockIdx.z) * 1048576)) + (ax0_ax1_fused_2_3 * 16384)) + ((((int)threadIdx.x) >> 4) * 8192)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)));
              int4 __34;
                int4 v__25 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                int4 v__26 = make_int4(2, 2, 2, 2);
                __34.x = (v__25.x%v__26.x);
                __34.y = (v__25.y%v__26.y);
                __34.z = (v__25.z%v__26.z);
                __34.w = (v__25.w%v__26.w);
              int4 __35;
                int4 v__27 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                int4 v__28 = make_int4(2, 2, 2, 2);
                __35.x = (v__27.x/v__28.x);
                __35.y = (v__27.y/v__28.y);
                __35.z = (v__27.z/v__28.z);
                __35.w = (v__27.w/v__28.w);
              int4 __36;
              ushort4 __37;
                ushort4 __38;
                  ushort4 __39;
                    int4 v__29 = make_int4(2, 2, 2, 2);
                    int4 v__30 = make_int4(0, 0, 0, 0);
                    __39.x = (v__29.x>=v__30.x);
                    __39.y = (v__29.y>=v__30.y);
                    __39.z = (v__29.z>=v__30.z);
                    __39.w = (v__29.w>=v__30.w);
                  ushort4 __40;
                    int4 v__31 = make_int4(0, 0, 0, 0);
                    __40.x = (__34.x>=v__31.x);
                    __40.y = (__34.y>=v__31.y);
                    __40.z = (__34.z>=v__31.z);
                    __40.w = (__34.w>=v__31.w);
                  __38.x = (__39.x&&__40.x);
                  __38.y = (__39.y&&__40.y);
                  __38.z = (__39.z&&__40.z);
                  __38.w = (__39.w&&__40.w);
                ushort4 __41;
                  ushort4 __42;
                    int4 v__32 = make_int4(2, 2, 2, 2);
                    int4 v__33 = make_int4(0, 0, 0, 0);
                    __42.x = (v__32.x<v__33.x);
                    __42.y = (v__32.y<v__33.y);
                    __42.z = (v__32.z<v__33.z);
                    __42.w = (v__32.w<v__33.w);
                  ushort4 __43;
                    int4 v__34 = make_int4(0, 0, 0, 0);
                    __43.x = (__34.x<=v__34.x);
                    __43.y = (__34.y<=v__34.y);
                    __43.z = (__34.z<=v__34.z);
                    __43.w = (__34.w<=v__34.w);
                  __41.x = (__42.x&&__43.x);
                  __41.y = (__42.y&&__43.y);
                  __41.z = (__42.z&&__43.z);
                  __41.w = (__42.w&&__43.w);
                __37.x = (__38.x||__41.x);
                __37.y = (__38.y||__41.y);
                __37.z = (__38.z||__41.z);
                __37.w = (__38.w||__41.w);
              int4 __44;
                int4 v__35 = make_int4(1, 1, 1, 1);
                __44.x = (__35.x-v__35.x);
                __44.y = (__35.y-v__35.y);
                __44.z = (__35.z-v__35.z);
                __44.w = (__35.w-v__35.w);
              __36.x = (bool(__37.x)?__35.x:__44.x);
              __36.y = (bool(__37.y)?__35.y:__44.y);
              __36.z = (bool(__37.z)?__35.z:__44.z);
              __36.w = (bool(__37.w)?__35.w:__44.w);
              __33.x = (v__24.x+__36.x);
              __33.y = (v__24.y+__36.y);
              __33.z = (v__24.z+__36.z);
              __33.w = (v__24.w+__36.w);
            int4 v__36 = make_int4(32, 32, 32, 32);
            __32.x = (__33.x+v__36.x);
            __32.y = (__33.y+v__36.y);
            __32.z = (__33.z+v__36.z);
            __32.w = (__33.w+v__36.w);
          int v__37 = ((0x000000ff << 0) & (B[__32.x] << 0))|((0x000000ff << 8) & (B[__32.y] << 8))|((0x000000ff << 16) & (B[__32.z] << 16))|((0x000000ff << 24) & (B[__32.w] << 24));
          __31.x = (int)(((char)(v__37 >> 0)));
          __31.y = (int)(((char)(v__37 >> 8)));
          __31.z = (int)(((char)(v__37 >> 16)));
          __31.w = (int)(((char)(v__37 >> 24)));
          int4 __45;
            int4 __46;
              int4 v__38 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
              int4 v__39 = make_int4(2, 2, 2, 2);
              __46.x = (v__38.x%v__39.x);
              __46.y = (v__38.y%v__39.y);
              __46.z = (v__38.z%v__39.z);
              __46.w = (v__38.w%v__39.w);
            int4 __47;
            ushort4 __48;
              ushort4 __49;
                ushort4 __50;
                  int4 v__40 = make_int4(2, 2, 2, 2);
                  int4 v__41 = make_int4(0, 0, 0, 0);
                  __50.x = (v__40.x>=v__41.x);
                  __50.y = (v__40.y>=v__41.y);
                  __50.z = (v__40.z>=v__41.z);
                  __50.w = (v__40.w>=v__41.w);
                ushort4 __51;
                  int4 v__42 = make_int4(0, 0, 0, 0);
                  __51.x = (__46.x>=v__42.x);
                  __51.y = (__46.y>=v__42.y);
                  __51.z = (__46.z>=v__42.z);
                  __51.w = (__46.w>=v__42.w);
                __49.x = (__50.x&&__51.x);
                __49.y = (__50.y&&__51.y);
                __49.z = (__50.z&&__51.z);
                __49.w = (__50.w&&__51.w);
              ushort4 __52;
                ushort4 __53;
                  int4 v__43 = make_int4(2, 2, 2, 2);
                  int4 v__44 = make_int4(0, 0, 0, 0);
                  __53.x = (v__43.x<v__44.x);
                  __53.y = (v__43.y<v__44.y);
                  __53.z = (v__43.z<v__44.z);
                  __53.w = (v__43.w<v__44.w);
                ushort4 __54;
                  int4 v__45 = make_int4(0, 0, 0, 0);
                  __54.x = (__46.x<=v__45.x);
                  __54.y = (__46.y<=v__45.y);
                  __54.z = (__46.z<=v__45.z);
                  __54.w = (__46.w<=v__45.w);
                __52.x = (__53.x&&__54.x);
                __52.y = (__53.y&&__54.y);
                __52.z = (__53.z&&__54.z);
                __52.w = (__53.w&&__54.w);
              __48.x = (__49.x||__52.x);
              __48.y = (__49.y||__52.y);
              __48.z = (__49.z||__52.z);
              __48.w = (__49.w||__52.w);
            int4 __55;
              int4 v__46 = make_int4(2, 2, 2, 2);
              __55.x = (__46.x+v__46.x);
              __55.y = (__46.y+v__46.y);
              __55.z = (__46.z+v__46.z);
              __55.w = (__46.w+v__46.w);
            __47.x = (bool(__48.x)?__46.x:__55.x);
            __47.y = (bool(__48.y)?__46.y:__55.y);
            __47.z = (bool(__48.z)?__46.z:__55.z);
            __47.w = (bool(__48.w)?__46.w:__55.w);
            int4 v__47 = make_int4(4, 4, 4, 4);
            __45.x = (__47.x*v__47.x);
            __45.y = (__47.y*v__47.y);
            __45.z = (__47.z*v__47.z);
            __45.w = (__47.w*v__47.w);
          __30.x = (__31.x >> __45.x);
          __30.y = (__31.y >> __45.y);
          __30.z = (__31.z >> __45.z);
          __30.w = (__31.w >> __45.w);
        int4 v__48 = make_int4(15, 15, 15, 15);
        __29.x = (__30.x & v__48.x);
        __29.y = (__30.y & v__48.y);
        __29.z = (__30.z & v__48.z);
        __29.w = (__30.w & v__48.w);
      ((half2*)(&(__28.x)))->x = (half)(__29.x);
      ((half2*)(&(__28.x)))->y = (half)(__29.y);
      ((half2*)(&(__28.y)))->x = (half)(__29.z);
      ((half2*)(&(__28.y)))->y = (half)(__29.w);
      *(uint2*)(B_decompress_shared + (((((k_0_0 + 1) & 1) * 8192) + (ax0_ax1_fused_2_3 * 128)) + (((int)threadIdx.x) * 4))) = __28;
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1) {
      nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[(((k_0_0 & 1) * 1024) + (k_0_1 * 16))])), 64);
      for (int ax0_0 = 0; ax0_0 < 8; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(B_decompress_shared_wmma_matrix_b[ax0_0], (&(B_decompress_shared[((((k_0_0 & 1) * 8192) + (ax0_0 * 1024)) + (k_0_1 * 16))])), 64);
      }
      for (int j_0_2 = 0; j_0_2 < 8; ++j_0_2) {
        nvcuda::wmma::mma_sync(C_wmma_accumulator[j_0_2], A_shared_wmma_matrix_a[0], B_decompress_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[j_0_2]);
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1) {
    nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((k_0_1_1 * 16) + 1024)])), 64);
    for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1) {
      nvcuda::wmma::load_matrix_sync(B_decompress_shared_wmma_matrix_b[ax0_0_1], (&(B_decompress_shared[(((ax0_0_1 * 1024) + (k_0_1_1 * 16)) + 8192)])), 64);
    }
    for (int j_0_2_1 = 0; j_0_2_1 < 8; ++j_0_2_1) {
      nvcuda::wmma::mma_sync(C_wmma_accumulator[j_0_2_1], A_shared_wmma_matrix_a[0], B_decompress_shared_wmma_matrix_b[j_0_2_1], C_wmma_accumulator[j_0_2_1]);
    }
  }
  for (int ax1_0 = 0; ax1_0 < 8; ++ax1_0) {
    nvcuda::wmma::store_matrix_sync((&(C[((((((int)blockIdx.y) * 262144) + (((int)blockIdx.x) * 1024)) + (((int)blockIdx.z) * 128)) + (ax1_0 * 16))])), C_wmma_accumulator[ax1_0], 16384, nvcuda::wmma::mem_row_major);
  }
}

