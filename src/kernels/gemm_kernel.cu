
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
using uint = unsigned int;
using uchar = unsigned char;
using ushort = unsigned short;
using int64_t = long long;
using uint64_t = unsigned long long;
#else
#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long
#endif

__global__ void __launch_bounds__(128) tir_halfxint3_tensorop_16x64x64x3_t0_y1z4_K9216_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[1];
    __shared__ half A_shared[3456];
    __shared__ half B_rescale_shared[13824];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[1];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[0], 0.000000e+00f);

    {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
        addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
        __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)(A_shared + (((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
        __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
            "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
            "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
            ::"r"(addr),
            "l"((void *)(A + ((((((int)blockIdx.y) * 147456) + (((int)threadIdx.z) * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16));
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 32; ++ax0_ax1_fused_2)
    {
        B_rescale_shared[((((((int)threadIdx.z) * 1152) + ((ax0_ax1_fused_2 >> 1) * 72)) + ((ax0_ax1_fused_2 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[(((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2 >> 1) * 3456)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2 >> 1) * 3456)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2 >> 1) * 3456)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 >> 1))]) - Zeros[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
        addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)) + 1152))));
#else
        __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)(A_shared + ((((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)) + 1152))));
#endif
        __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
            "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
            "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
            ::"r"(addr),
            "l"((void *)(A + (((((((int)blockIdx.y) * 147456) + (((int)threadIdx.z) * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16));
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 32; ++ax0_ax1_fused_2_1)
    {
        B_rescale_shared[(((((((int)threadIdx.z) * 1152) + ((ax0_ax1_fused_2_1 >> 1) * 72)) + ((ax0_ax1_fused_2_1 & 1) * 32)) + ((int)threadIdx.x)) + 4608)] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 >> 1))]) - Zeros[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 142; ++k_0_0)
    {
        __syncthreads();

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((k_0_0 + 2) % 3) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + ((((((k_0_0 + 2) % 3) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + ((((((((int)blockIdx.y) * 147456) + (((int)threadIdx.z) * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 128))), "n"(16));
        }
        for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 32; ++ax0_ax1_fused_2_2)
        {
            B_rescale_shared[(((((((k_0_0 + 2) % 3) * 4608) + (((int)threadIdx.z) * 1152)) + ((ax0_ax1_fused_2_2 >> 1) * 72)) + ((ax0_ax1_fused_2_2 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[(((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2_2 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 48)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2_2 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 48)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.z) * 55296)) + ((ax0_ax1_fused_2_2 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 49)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_2 >> 1))]) - Zeros[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_2 >> 1))]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 2;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[(((k_0_0 % 3) * 1152) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 % 3) * 4608) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
        }
    }
    __asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1)
    {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((k_0_1_1 * 16) + 1152)])), 72);
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 4608)])), 72);
        nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_2 = 0; k_0_1_2 < 4; ++k_0_1_2)
    {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((k_0_1_2 * 16) + 2304)])), 72);
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_2 * 16)) + 9216)])), 72);
        nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
    }
    nvcuda::wmma::store_matrix_sync((&(C[(((((int)blockIdx.y) * 147456) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[0], 9216, nvcuda::wmma::mem_row_major);
}

__global__ void __launch_bounds__(256) tir_halfxint3_tensorop_32x64x64x2_t0_y2z4_K9216_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[1];
    __shared__ half A_shared[4608];
    __shared__ half B_rescale_shared[9216];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[1];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[0], 0.000000e+00f);

    {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
        addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((int)threadIdx.y) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
        __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)(A_shared + ((((((int)threadIdx.y) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
        __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
            "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
            "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
            ::"r"(addr),
            "l"((void *)(A + (((((((int)blockIdx.y) * 294912) + (((int)threadIdx.y) * 147456)) + (((int)threadIdx.z) * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16));
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 16; ++ax0_ax1_fused_2)
    {
        B_rescale_shared[(((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2 >> 1) * 72)) + ((ax0_ax1_fused_2 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2 >> 1) * 3456)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2 >> 1) * 3456)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2 >> 1) * 3456)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 143; ++k_0_0)
    {
        __syncthreads();

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((k_0_0 + 1) & 1) * 2304) + (((int)threadIdx.y) * 1152)) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + (((((((k_0_0 + 1) & 1) * 2304) + (((int)threadIdx.y) * 1152)) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + (((((((((int)blockIdx.y) * 294912) + (((int)threadIdx.y) * 147456)) + (((int)threadIdx.z) * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16));
        }
        for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 16; ++ax0_ax1_fused_2_1)
        {
            B_rescale_shared[((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2_1 >> 1) * 72)) + ((ax0_ax1_fused_2_1 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 1;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((((k_0_0 & 1) * 2304) + (((int)threadIdx.y) * 1152)) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 & 1) * 4608) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
        }
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1)
    {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[(((((int)threadIdx.y) * 1152) + (k_0_1_1 * 16)) + 2304)])), 72);
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 4608)])), 72);
        nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
    }
    nvcuda::wmma::store_matrix_sync((&(C[((((((int)blockIdx.y) * 294912) + (((int)threadIdx.y) * 147456)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[0], 9216, nvcuda::wmma::mem_row_major);
}

__global__ void __launch_bounds__(256) tir_halfxint3_tensorop_64x64x64x2_t0_y2z4_K9216_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[2];
    __shared__ half A_shared[9216];
    __shared__ half B_rescale_shared[9216];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[2];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
    for (int i_0_2_init = 0; i_0_2_init < 2; ++i_0_2_init)
    {
        nvcuda::wmma::fill_fragment(C_wmma_accumulator[i_0_2_init], 0.000000e+00f);
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 2; ++ax0_ax1_fused_2)
    {

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + (((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 294912)) + (((int)threadIdx.z) * 73728)) + (ax0_ax1_fused_2 * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16));
        }
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 16; ++ax0_ax1_fused_2_1)
    {
        B_rescale_shared[(((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2_1 >> 1) * 72)) + ((ax0_ax1_fused_2_1 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_1 >> 1) * 3456)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 143; ++k_0_0)
    {
        __syncthreads();
        for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 2; ++ax0_ax1_fused_2_2)
        {

            {
                unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
                addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
                __asm__ __volatile__(
                    "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                    : "=r"(addr)
                    : "l"((void *)(A_shared + ((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
                __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                    "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                    "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                    ::"r"(addr),
                    "l"((void *)(A + ((((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 294912)) + (((int)threadIdx.z) * 73728)) + (ax0_ax1_fused_2_2 * 36864)) + ((((int)threadIdx.x) >> 3) * 9216)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16));
            }
        }
        for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 16; ++ax0_ax1_fused_2_3)
        {
            B_rescale_shared[((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2_3 >> 1) * 72)) + ((ax0_ax1_fused_2_3 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_3 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_3 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_3 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_3 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((((int)blockIdx.x) * 221184) + (((int)threadIdx.y) * 110592)) + (((int)threadIdx.z) * 27648)) + ((ax0_ax1_fused_2_3 >> 1) * 3456)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_3 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_3 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_3 >> 1))]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 1;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1)
        {
            for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0)
            {
                nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0], (&(A_shared[(((((k_0_0 & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (ax0_0 * 1152)) + (k_0_1 * 16))])), 72);
            }
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 & 1) * 4608) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
            for (int i_0_2 = 0; i_0_2 < 2; ++i_0_2)
            {
                nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2], A_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2]);
            }
        }
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1)
    {
        for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0_1], (&(A_shared[((((((int)threadIdx.y) * 2304) + (ax0_0_1 * 1152)) + (k_0_1_1 * 16)) + 4608)])), 72);
        }
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 4608)])), 72);
        for (int i_0_2_1 = 0; i_0_2_1 < 2; ++i_0_2_1)
        {
            nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2_1], A_shared_wmma_matrix_a[i_0_2_1], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2_1]);
        }
    }
    for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2)
    {
        nvcuda::wmma::store_matrix_sync((&(C[(((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 294912)) + (ax0_0_2 * 147456)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[ax0_0_2], 9216, nvcuda::wmma::mem_row_major);
    }
}

__global__ void __launch_bounds__(64) tir_halfxint3_tensorop_64x128x32x3_t0_y1z2_K9216_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[16];
    __shared__ half A_shared[7680];
    __shared__ half B_rescale_shared[15360];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[4];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[4];
    for (int i_0_2_init = 0; i_0_2_init < 4; ++i_0_2_init)
    {
        for (int j_0_2_init = 0; j_0_2_init < 4; ++j_0_2_init)
        {
            nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_0_2_init * 4) + j_0_2_init)], 0.000000e+00f);
        }
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2)
    {

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + ((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + (((((((int)blockIdx.y) * 589824) + (((int)threadIdx.z) * 294912)) + (ax0_ax1_fused_2 * 73728)) + ((((int)threadIdx.x) >> 2) * 9216)) + ((((int)threadIdx.x) & 3) * 8)))), "n"(16));
        }
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 64; ++ax0_ax1_fused_2_1)
    {
        B_rescale_shared[(((((int)threadIdx.z) * 2560) + (ax0_ax1_fused_2_1 * 40)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_1 * 3456)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_1 * 3456)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_1 * 3456)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]) - Zeros[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 4; ++ax0_ax1_fused_2_2)
    {

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 2560))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + (((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 2560))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.z) * 294912)) + (ax0_ax1_fused_2_2 * 73728)) + ((((int)threadIdx.x) >> 2) * 9216)) + ((((int)threadIdx.x) & 3) * 8)) + 32))), "n"(16));
        }
    }
    for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 64; ++ax0_ax1_fused_2_3)
    {
        B_rescale_shared[((((((int)threadIdx.z) * 2560) + (ax0_ax1_fused_2_3 * 40)) + ((int)threadIdx.x)) + 5120)] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[(((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_3 * 3456)) + ((((int)threadIdx.x) * 3) >> 3)) + 12)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_3 * 3456)) + ((((int)threadIdx.x) * 3) >> 3)) + 12)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_3 * 3456)) + ((((int)threadIdx.x) * 3) >> 3)) + 13)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_3)]) - Zeros[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_3)]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 286; ++k_0_0)
    {
        __syncthreads();
        for (int ax0_ax1_fused_2_4 = 0; ax0_ax1_fused_2_4 < 4; ++ax0_ax1_fused_2_4)
        {

            {
                unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
                addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((k_0_0 + 2) % 3) * 2560) + (((int)threadIdx.z) * 1280)) + (ax0_ax1_fused_2_4 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#else
                __asm__ __volatile__(
                    "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                    : "=r"(addr)
                    : "l"((void *)(A_shared + (((((((k_0_0 + 2) % 3) * 2560) + (((int)threadIdx.z) * 1280)) + (ax0_ax1_fused_2_4 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#endif
                __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                    "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                    "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                    ::"r"(addr),
                    "l"((void *)(A + (((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.z) * 294912)) + (ax0_ax1_fused_2_4 * 73728)) + ((((int)threadIdx.x) >> 2) * 9216)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 64))), "n"(16));
            }
        }
        for (int ax0_ax1_fused_2_5 = 0; ax0_ax1_fused_2_5 < 64; ++ax0_ax1_fused_2_5)
        {
            B_rescale_shared[((((((k_0_0 + 2) % 3) * 5120) + (((int)threadIdx.z) * 2560)) + (ax0_ax1_fused_2_5 * 40)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_5 * 3456)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_5 * 3456)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 442368) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_5 * 3456)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_5)]) - Zeros[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_5)]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 2;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1)
        {
            for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0)
            {
                nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0], (&(A_shared[((((k_0_0 % 3) * 2560) + (ax0_0 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int ax0_0_1 = 0; ax0_0_1 < 4; ++ax0_0_1)
            {
                nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_1], (&(B_rescale_shared[(((((k_0_0 % 3) * 5120) + (((int)threadIdx.z) * 2560)) + (ax0_0_1 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int i_0_2 = 0; i_0_2 < 4; ++i_0_2)
            {
                for (int j_0_2 = 0; j_0_2 < 4; ++j_0_2)
                {
                    nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2 * 4) + j_0_2)], A_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[((i_0_2 * 4) + j_0_2)]);
                }
            }
        }
    }
    __asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 2; ++k_0_1_1)
    {
        for (int ax0_0_2 = 0; ax0_0_2 < 4; ++ax0_0_2)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0_2], (&(A_shared[(((ax0_0_2 * 640) + (k_0_1_1 * 16)) + 2560)])), 40);
        }
        for (int ax0_0_3 = 0; ax0_0_3 < 4; ++ax0_0_3)
        {
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_3], (&(B_rescale_shared[((((((int)threadIdx.z) * 2560) + (ax0_0_3 * 640)) + (k_0_1_1 * 16)) + 5120)])), 40);
        }
        for (int i_0_2_1 = 0; i_0_2_1 < 4; ++i_0_2_1)
        {
            for (int j_0_2_1 = 0; j_0_2_1 < 4; ++j_0_2_1)
            {
                nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2_1 * 4) + j_0_2_1)], A_shared_wmma_matrix_a[i_0_2_1], B_rescale_shared_wmma_matrix_b[j_0_2_1], C_wmma_accumulator[((i_0_2_1 * 4) + j_0_2_1)]);
            }
        }
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_2 = 0; k_0_1_2 < 2; ++k_0_1_2)
    {
        for (int ax0_0_4 = 0; ax0_0_4 < 4; ++ax0_0_4)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0_4], (&(A_shared[(((ax0_0_4 * 640) + (k_0_1_2 * 16)) + 5120)])), 40);
        }
        for (int ax0_0_5 = 0; ax0_0_5 < 4; ++ax0_0_5)
        {
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_5], (&(B_rescale_shared[((((((int)threadIdx.z) * 2560) + (ax0_0_5 * 640)) + (k_0_1_2 * 16)) + 10240)])), 40);
        }
        for (int i_0_2_2 = 0; i_0_2_2 < 4; ++i_0_2_2)
        {
            for (int j_0_2_2 = 0; j_0_2_2 < 4; ++j_0_2_2)
            {
                nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2_2 * 4) + j_0_2_2)], A_shared_wmma_matrix_a[i_0_2_2], B_rescale_shared_wmma_matrix_b[j_0_2_2], C_wmma_accumulator[((i_0_2_2 * 4) + j_0_2_2)]);
            }
        }
    }
    for (int ax0_0_6 = 0; ax0_0_6 < 4; ++ax0_0_6)
    {
        for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0)
        {
            nvcuda::wmma::store_matrix_sync((&(C[(((((((int)blockIdx.y) * 589824) + (ax0_0_6 * 147456)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.z) * 64)) + (ax1_0 * 16))])), C_wmma_accumulator[((ax0_0_6 * 4) + ax1_0)], 9216, nvcuda::wmma::mem_row_major);
        }
    }
}

__global__ void __launch_bounds__(128) tir_halfxint3_tensorop_128x256x32x1_t0_y2z2_K9216_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[32];
    __shared__ half A_shared[5120];
    __shared__ half B_rescale_shared[10240];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[4];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[8];
    for (int i_0_2_init = 0; i_0_2_init < 4; ++i_0_2_init)
    {
        for (int j_0_2_init = 0; j_0_2_init < 8; ++j_0_2_init)
        {
            nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_0_2_init * 8) + j_0_2_init)], 0.000000e+00f);
        }
    }
    for (int k_0_0 = 0; k_0_0 < 288; ++k_0_0)
    {
        __syncthreads();
        for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2)
        {
            *(uint4 *)(A_shared + (((((((int)threadIdx.y) * 2560) + (((int)threadIdx.z) * 1280)) + (ax0_ax1_fused_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8))) = *(uint4 *)(A + (((((((((int)blockIdx.y) * 1179648) + (((int)threadIdx.y) * 589824)) + (((int)threadIdx.z) * 294912)) + (ax0_ax1_fused_2 * 73728)) + ((((int)threadIdx.x) >> 2) * 9216)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)));
        }
        for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 64; ++ax0_ax1_fused_2_1)
        {
            B_rescale_shared[((((((int)threadIdx.y) * 5120) + (((int)threadIdx.z) * 2560)) + (ax0_ax1_fused_2_1 * 40)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_1 * 3456)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_1 * 3456)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 221184)) + (ax0_ax1_fused_2_1 * 3456)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 256) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]) - Zeros[((((((int)blockIdx.x) * 256) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]);
        }
        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1)
        {
            for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0)
            {
                nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0], (&(A_shared[(((((int)threadIdx.y) * 2560) + (ax0_0 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1)
            {
                nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_1], (&(B_rescale_shared[(((((int)threadIdx.z) * 5120) + (ax0_0_1 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int i_0_2 = 0; i_0_2 < 4; ++i_0_2)
            {
                for (int j_0_2 = 0; j_0_2 < 8; ++j_0_2)
                {
                    nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2 * 8) + j_0_2)], A_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[((i_0_2 * 8) + j_0_2)]);
                }
            }
        }
    }
    for (int ax0_0_2 = 0; ax0_0_2 < 4; ++ax0_0_2)
    {
        for (int ax1_0 = 0; ax1_0 < 8; ++ax1_0)
        {
            nvcuda::wmma::store_matrix_sync((&(C[((((((((int)blockIdx.y) * 1179648) + (((int)threadIdx.y) * 589824)) + (ax0_0_2 * 147456)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.z) * 128)) + (ax1_0 * 16))])), C_wmma_accumulator[((ax0_0_2 * 8) + ax1_0)], 9216, nvcuda::wmma::mem_row_major);
        }
    }
}

__global__ void __launch_bounds__(128) tir_halfxint3_tensorop_16x64x64x3_t0_y1z4_K36864_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[1];
    __shared__ half A_shared[3456];
    __shared__ half B_rescale_shared[13824];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[1];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[0], 0.000000e+00f);

    {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
        addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
        __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)(A_shared + (((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
        __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
            "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
            "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
            ::"r"(addr),
            "l"((void *)(A + ((((((int)blockIdx.y) * 589824) + (((int)threadIdx.z) * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16));
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 32; ++ax0_ax1_fused_2)
    {
        B_rescale_shared[((((((int)threadIdx.z) * 1152) + ((ax0_ax1_fused_2 >> 1) * 72)) + ((ax0_ax1_fused_2 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[(((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2 >> 1) * 13824)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2 >> 1) * 13824)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2 >> 1) * 13824)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 >> 1))]) - Zeros[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
        addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)) + 1152))));
#else
        __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)(A_shared + ((((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)) + 1152))));
#endif
        __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
            "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
            "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
            ::"r"(addr),
            "l"((void *)(A + (((((((int)blockIdx.y) * 589824) + (((int)threadIdx.z) * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16));
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 32; ++ax0_ax1_fused_2_1)
    {
        B_rescale_shared[(((((((int)threadIdx.z) * 1152) + ((ax0_ax1_fused_2_1 >> 1) * 72)) + ((ax0_ax1_fused_2_1 & 1) * 32)) + ((int)threadIdx.x)) + 4608)] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 >> 1))]) - Zeros[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 574; ++k_0_0)
    {
        __syncthreads();

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((k_0_0 + 2) % 3) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + ((((((k_0_0 + 2) % 3) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.z) * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 128))), "n"(16));
        }
        for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 32; ++ax0_ax1_fused_2_2)
        {
            B_rescale_shared[(((((((k_0_0 + 2) % 3) * 4608) + (((int)threadIdx.z) * 1152)) + ((ax0_ax1_fused_2_2 >> 1) * 72)) + ((ax0_ax1_fused_2_2 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[(((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2_2 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 48)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2_2 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 48)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.z) * 221184)) + ((ax0_ax1_fused_2_2 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 49)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_2 >> 1))]) - Zeros[(((((int)blockIdx.x) * 64) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_2 >> 1))]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 2;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[(((k_0_0 % 3) * 1152) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 % 3) * 4608) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
        }
    }
    __asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1)
    {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((k_0_1_1 * 16) + 1152)])), 72);
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 4608)])), 72);
        nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_2 = 0; k_0_1_2 < 4; ++k_0_1_2)
    {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((k_0_1_2 * 16) + 2304)])), 72);
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_2 * 16)) + 9216)])), 72);
        nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
    }
    nvcuda::wmma::store_matrix_sync((&(C[(((((int)blockIdx.y) * 147456) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[0], 9216, nvcuda::wmma::mem_row_major);
}

__global__ void __launch_bounds__(256) tir_halfxint3_tensorop_32x64x64x2_t0_y2z4_K36864_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[1];
    __shared__ half A_shared[4608];
    __shared__ half B_rescale_shared[9216];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[1];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[0], 0.000000e+00f);

    {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
        addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((int)threadIdx.y) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
        __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)(A_shared + ((((((int)threadIdx.y) * 1152) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
        __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
            "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
            "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
            ::"r"(addr),
            "l"((void *)(A + (((((((int)blockIdx.y) * 1179648) + (((int)threadIdx.y) * 589824)) + (((int)threadIdx.z) * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16));
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 16; ++ax0_ax1_fused_2)
    {
        B_rescale_shared[(((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2 >> 1) * 72)) + ((ax0_ax1_fused_2 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2 >> 1) * 13824)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2 >> 1) * 13824)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2 >> 1) * 13824)) + ((ax0_ax1_fused_2 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 575; ++k_0_0)
    {
        __syncthreads();

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((k_0_0 + 1) & 1) * 2304) + (((int)threadIdx.y) * 1152)) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + (((((((k_0_0 + 1) & 1) * 2304) + (((int)threadIdx.y) * 1152)) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + (((((((((int)blockIdx.y) * 1179648) + (((int)threadIdx.y) * 589824)) + (((int)threadIdx.z) * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16));
        }
        for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 16; ++ax0_ax1_fused_2_1)
        {
            B_rescale_shared[((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2_1 >> 1) * 72)) + ((ax0_ax1_fused_2_1 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 1;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[((((k_0_0 & 1) * 2304) + (((int)threadIdx.y) * 1152)) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 & 1) * 4608) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
            nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
        }
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1)
    {
        nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[0], (&(A_shared[(((((int)threadIdx.y) * 1152) + (k_0_1_1 * 16)) + 2304)])), 72);
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 4608)])), 72);
        nvcuda::wmma::mma_sync(C_wmma_accumulator[0], A_shared_wmma_matrix_a[0], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[0]);
    }
    nvcuda::wmma::store_matrix_sync((&(C[((((((int)blockIdx.y) * 294912) + (((int)threadIdx.y) * 147456)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[0], 9216, nvcuda::wmma::mem_row_major);
}

__global__ void __launch_bounds__(256) tir_halfxint3_tensorop_64x64x64x2_t0_y2z4_K36864_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[2];
    __shared__ half A_shared[9216];
    __shared__ half B_rescale_shared[9216];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[2];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
    for (int i_0_2_init = 0; i_0_2_init < 2; ++i_0_2_init)
    {
        nvcuda::wmma::fill_fragment(C_wmma_accumulator[i_0_2_init], 0.000000e+00f);
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 2; ++ax0_ax1_fused_2)
    {

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + (((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + ((((((((int)blockIdx.y) * 2359296) + (((int)threadIdx.y) * 1179648)) + (((int)threadIdx.z) * 294912)) + (ax0_ax1_fused_2 * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16));
        }
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 16; ++ax0_ax1_fused_2_1)
    {
        B_rescale_shared[(((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2_1 >> 1) * 72)) + ((ax0_ax1_fused_2_1 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_1 >> 1) * 13824)) + ((ax0_ax1_fused_2_1 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_1 >> 1))]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 575; ++k_0_0)
    {
        __syncthreads();
        for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 2; ++ax0_ax1_fused_2_2)
        {

            {
                unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
                addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#else
                __asm__ __volatile__(
                    "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                    : "=r"(addr)
                    : "l"((void *)(A_shared + ((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2_2 * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8)))));
#endif
                __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                    "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                    "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                    ::"r"(addr),
                    "l"((void *)(A + ((((((((((int)blockIdx.y) * 2359296) + (((int)threadIdx.y) * 1179648)) + (((int)threadIdx.z) * 294912)) + (ax0_ax1_fused_2_2 * 147456)) + ((((int)threadIdx.x) >> 3) * 36864)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16));
            }
        }
        for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 16; ++ax0_ax1_fused_2_3)
        {
            B_rescale_shared[((((((((k_0_0 + 1) & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.z) * 576)) + ((ax0_ax1_fused_2_3 >> 1) * 72)) + ((ax0_ax1_fused_2_3 & 1) * 32)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_3 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_3 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_3 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_3 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((((int)blockIdx.x) * 884736) + (((int)threadIdx.y) * 442368)) + (((int)threadIdx.z) * 110592)) + ((ax0_ax1_fused_2_3 >> 1) * 13824)) + (k_0_0 * 24)) + ((ax0_ax1_fused_2_3 & 1) * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_3 >> 1))]) - Zeros[((((((int)blockIdx.x) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2_3 >> 1))]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 1;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1)
        {
            for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0)
            {
                nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0], (&(A_shared[(((((k_0_0 & 1) * 4608) + (((int)threadIdx.y) * 2304)) + (ax0_0 * 1152)) + (k_0_1 * 16))])), 72);
            }
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 & 1) * 4608) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
            for (int i_0_2 = 0; i_0_2 < 2; ++i_0_2)
            {
                nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2], A_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2]);
            }
        }
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1)
    {
        for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0_1], (&(A_shared[((((((int)threadIdx.y) * 2304) + (ax0_0_1 * 1152)) + (k_0_1_1 * 16)) + 4608)])), 72);
        }
        nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 4608)])), 72);
        for (int i_0_2_1 = 0; i_0_2_1 < 2; ++i_0_2_1)
        {
            nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2_1], A_shared_wmma_matrix_a[i_0_2_1], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2_1]);
        }
    }
    for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2)
    {
        nvcuda::wmma::store_matrix_sync((&(C[(((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 294912)) + (ax0_0_2 * 147456)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[ax0_0_2], 9216, nvcuda::wmma::mem_row_major);
    }
}

__global__ void __launch_bounds__(64) tir_halfxint3_tensorop_64x128x32x3_t0_y1z2_K36864_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[16];
    __shared__ half A_shared[7680];
    __shared__ half B_rescale_shared[15360];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[4];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[4];
    for (int i_0_2_init = 0; i_0_2_init < 4; ++i_0_2_init)
    {
        for (int j_0_2_init = 0; j_0_2_init < 4; ++j_0_2_init)
        {
            nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_0_2_init * 4) + j_0_2_init)], 0.000000e+00f);
        }
    }
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2)
    {

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + ((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + (((((((int)blockIdx.y) * 2359296) + (((int)threadIdx.z) * 1179648)) + (ax0_ax1_fused_2 * 294912)) + ((((int)threadIdx.x) >> 2) * 36864)) + ((((int)threadIdx.x) & 3) * 8)))), "n"(16));
        }
    }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 64; ++ax0_ax1_fused_2_1)
    {
        B_rescale_shared[(((((int)threadIdx.z) * 2560) + (ax0_ax1_fused_2_1 * 40)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_1 * 13824)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_1 * 13824)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_1 * 13824)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]) - Zeros[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 4; ++ax0_ax1_fused_2_2)
    {

        {
            unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
            addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 2560))));
#else
            __asm__ __volatile__(
                "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                : "=r"(addr)
                : "l"((void *)(A_shared + (((((((int)threadIdx.z) * 1280) + (ax0_ax1_fused_2_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 2560))));
#endif
            __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                ::"r"(addr),
                "l"((void *)(A + ((((((((int)blockIdx.y) * 2359296) + (((int)threadIdx.z) * 1179648)) + (ax0_ax1_fused_2_2 * 294912)) + ((((int)threadIdx.x) >> 2) * 36864)) + ((((int)threadIdx.x) & 3) * 8)) + 32))), "n"(16));
        }
    }
    for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 64; ++ax0_ax1_fused_2_3)
    {
        B_rescale_shared[((((((int)threadIdx.z) * 2560) + (ax0_ax1_fused_2_3 * 40)) + ((int)threadIdx.x)) + 5120)] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[(((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_3 * 13824)) + ((((int)threadIdx.x) * 3) >> 3)) + 12)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_3 * 13824)) + ((((int)threadIdx.x) * 3) >> 3)) + 12)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_3 * 13824)) + ((((int)threadIdx.x) * 3) >> 3)) + 13)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_3)]) - Zeros[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_3)]);
    }
    __asm__ __volatile__("cp.async.commit_group;");

    for (int k_0_0 = 0; k_0_0 < 1150; ++k_0_0)
    {
        __syncthreads();
        for (int ax0_ax1_fused_2_4 = 0; ax0_ax1_fused_2_4 < 4; ++ax0_ax1_fused_2_4)
        {

            {
                unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
                addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((((k_0_0 + 2) % 3) * 2560) + (((int)threadIdx.z) * 1280)) + (ax0_ax1_fused_2_4 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#else
                __asm__ __volatile__(
                    "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
                    : "=r"(addr)
                    : "l"((void *)(A_shared + (((((((k_0_0 + 2) % 3) * 2560) + (((int)threadIdx.z) * 1280)) + (ax0_ax1_fused_2_4 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)))));
#endif
                __asm__ __volatile__(
#if TVM_ENABLE_L2_PREFETCH
                    "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
#else
                    "cp.async.cg.shared.global [%0], [%1], %2;"
#endif
                    ::"r"(addr),
                    "l"((void *)(A + (((((((((int)blockIdx.y) * 2359296) + (((int)threadIdx.z) * 1179648)) + (ax0_ax1_fused_2_4 * 294912)) + ((((int)threadIdx.x) >> 2) * 36864)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 64))), "n"(16));
            }
        }
        for (int ax0_ax1_fused_2_5 = 0; ax0_ax1_fused_2_5 < 64; ++ax0_ax1_fused_2_5)
        {
            B_rescale_shared[((((((k_0_0 + 2) % 3) * 5120) + (((int)threadIdx.z) * 2560)) + (ax0_ax1_fused_2_5 * 40)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_5 * 13824)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_5 * 13824)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 24)]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 1769472) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_5 * 13824)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 25)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_5)]) - Zeros[(((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_5)]);
        }
        __asm__ __volatile__("cp.async.commit_group;");

        __asm__ __volatile__("cp.async.wait_group 2;");

        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1)
        {
            for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0)
            {
                nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0], (&(A_shared[((((k_0_0 % 3) * 2560) + (ax0_0 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int ax0_0_1 = 0; ax0_0_1 < 4; ++ax0_0_1)
            {
                nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_1], (&(B_rescale_shared[(((((k_0_0 % 3) * 5120) + (((int)threadIdx.z) * 2560)) + (ax0_0_1 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int i_0_2 = 0; i_0_2 < 4; ++i_0_2)
            {
                for (int j_0_2 = 0; j_0_2 < 4; ++j_0_2)
                {
                    nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2 * 4) + j_0_2)], A_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[((i_0_2 * 4) + j_0_2)]);
                }
            }
        }
    }
    __asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int k_0_1_1 = 0; k_0_1_1 < 2; ++k_0_1_1)
    {
        for (int ax0_0_2 = 0; ax0_0_2 < 4; ++ax0_0_2)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0_2], (&(A_shared[(((ax0_0_2 * 640) + (k_0_1_1 * 16)) + 2560)])), 40);
        }
        for (int ax0_0_3 = 0; ax0_0_3 < 4; ++ax0_0_3)
        {
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_3], (&(B_rescale_shared[((((((int)threadIdx.z) * 2560) + (ax0_0_3 * 640)) + (k_0_1_1 * 16)) + 5120)])), 40);
        }
        for (int i_0_2_1 = 0; i_0_2_1 < 4; ++i_0_2_1)
        {
            for (int j_0_2_1 = 0; j_0_2_1 < 4; ++j_0_2_1)
            {
                nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2_1 * 4) + j_0_2_1)], A_shared_wmma_matrix_a[i_0_2_1], B_rescale_shared_wmma_matrix_b[j_0_2_1], C_wmma_accumulator[((i_0_2_1 * 4) + j_0_2_1)]);
            }
        }
    }
    __asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int k_0_1_2 = 0; k_0_1_2 < 2; ++k_0_1_2)
    {
        for (int ax0_0_4 = 0; ax0_0_4 < 4; ++ax0_0_4)
        {
            nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0_4], (&(A_shared[(((ax0_0_4 * 640) + (k_0_1_2 * 16)) + 5120)])), 40);
        }
        for (int ax0_0_5 = 0; ax0_0_5 < 4; ++ax0_0_5)
        {
            nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_5], (&(B_rescale_shared[((((((int)threadIdx.z) * 2560) + (ax0_0_5 * 640)) + (k_0_1_2 * 16)) + 10240)])), 40);
        }
        for (int i_0_2_2 = 0; i_0_2_2 < 4; ++i_0_2_2)
        {
            for (int j_0_2_2 = 0; j_0_2_2 < 4; ++j_0_2_2)
            {
                nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2_2 * 4) + j_0_2_2)], A_shared_wmma_matrix_a[i_0_2_2], B_rescale_shared_wmma_matrix_b[j_0_2_2], C_wmma_accumulator[((i_0_2_2 * 4) + j_0_2_2)]);
            }
        }
    }
    for (int ax0_0_6 = 0; ax0_0_6 < 4; ++ax0_0_6)
    {
        for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0)
        {
            nvcuda::wmma::store_matrix_sync((&(C[(((((((int)blockIdx.y) * 589824) + (ax0_0_6 * 147456)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.z) * 64)) + (ax1_0 * 16))])), C_wmma_accumulator[((ax0_0_6 * 4) + ax1_0)], 9216, nvcuda::wmma::mem_row_major);
        }
    }
}

__global__ void __launch_bounds__(128) tir_halfxint3_tensorop_128x256x32x1_t0_y2z2_K36864_align8(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[32];
    __shared__ half A_shared[5120];
    __shared__ half B_rescale_shared[10240];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_shared_wmma_matrix_a[4];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[8];
    for (int i_0_2_init = 0; i_0_2_init < 4; ++i_0_2_init)
    {
        for (int j_0_2_init = 0; j_0_2_init < 8; ++j_0_2_init)
        {
            nvcuda::wmma::fill_fragment(C_wmma_accumulator[((i_0_2_init * 8) + j_0_2_init)], 0.000000e+00f);
        }
    }
    for (int k_0_0 = 0; k_0_0 < 1152; ++k_0_0)
    {
        __syncthreads();
        for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2)
        {
            *(uint4 *)(A_shared + (((((((int)threadIdx.y) * 2560) + (((int)threadIdx.z) * 1280)) + (ax0_ax1_fused_2 * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8))) = *(uint4 *)(A + (((((((((int)blockIdx.y) * 4718592) + (((int)threadIdx.y) * 2359296)) + (((int)threadIdx.z) * 1179648)) + (ax0_ax1_fused_2 * 294912)) + ((((int)threadIdx.x) >> 2) * 36864)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)));
        }
        for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 64; ++ax0_ax1_fused_2_1)
        {
            B_rescale_shared[((((((int)threadIdx.y) * 5120) + (((int)threadIdx.z) * 2560)) + (ax0_ax1_fused_2_1 * 40)) + ((int)threadIdx.x))] = ((((((((int)threadIdx.x) * 3) & 7) <= 5) ? ((half)((((int)B[((((((((int)blockIdx.x) * 3538944) + (((int)threadIdx.y) * 1769472)) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_1 * 13824)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[((((((((int)blockIdx.x) * 3538944) + (((int)threadIdx.y) * 1769472)) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_1 * 13824)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3))]) >> ((((int)threadIdx.x) * 3) & 7)) & ((1 << (8 - ((((int)threadIdx.x) * 3) & 7))) - 1))) | ((signed char)(((((int)B[(((((((((int)blockIdx.x) * 3538944) + (((int)threadIdx.y) * 1769472)) + (((int)threadIdx.z) * 884736)) + (ax0_ax1_fused_2_1 * 13824)) + (k_0_0 * 12)) + ((((int)threadIdx.x) * 3) >> 3)) + 1)]) << (8 - ((((int)threadIdx.x) * 3) & 7))) & (7 << (8 - ((((int)threadIdx.x) * 3) & 7)))) & 7))))) * Scales[((((((int)blockIdx.x) * 256) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]) - Zeros[((((((int)blockIdx.x) * 256) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.z) * 64)) + ax0_ax1_fused_2_1)]);
        }
        __syncthreads();
        for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1)
        {
            for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0)
            {
                nvcuda::wmma::load_matrix_sync(A_shared_wmma_matrix_a[ax0_0], (&(A_shared[(((((int)threadIdx.y) * 2560) + (ax0_0 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int ax0_0_1 = 0; ax0_0_1 < 8; ++ax0_0_1)
            {
                nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[ax0_0_1], (&(B_rescale_shared[(((((int)threadIdx.z) * 5120) + (ax0_0_1 * 640)) + (k_0_1 * 16))])), 40);
            }
            for (int i_0_2 = 0; i_0_2 < 4; ++i_0_2)
            {
                for (int j_0_2 = 0; j_0_2 < 8; ++j_0_2)
                {
                    nvcuda::wmma::mma_sync(C_wmma_accumulator[((i_0_2 * 8) + j_0_2)], A_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[j_0_2], C_wmma_accumulator[((i_0_2 * 8) + j_0_2)]);
                }
            }
        }
    }
    for (int ax0_0_2 = 0; ax0_0_2 < 4; ++ax0_0_2)
    {
        for (int ax1_0 = 0; ax1_0 < 8; ++ax1_0)
        {
            nvcuda::wmma::store_matrix_sync((&(C[((((((((int)blockIdx.y) * 1179648) + (((int)threadIdx.y) * 589824)) + (ax0_0_2 * 147456)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.z) * 128)) + (ax1_0 * 16))])), C_wmma_accumulator[((ax0_0_2 * 8) + ax1_0)], 9216, nvcuda::wmma::mem_row_major);
        }
    }
}
