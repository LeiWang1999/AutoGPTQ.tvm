
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
using uint = unsigned int;
using uchar = unsigned char;
using ushort = unsigned short;
using int64_t = long long;
using uint64_t = unsigned long long;
#else
#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long
#endif

extern "C" __global__ void __launch_bounds__(128) tir_halfxint3_simt_bn4_k9216(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    half in_thread_C_local[1];
    half A_local[8];
    half red_buf0[1];
    in_thread_C_local[0] = __float2half_rn(0.000000e+00f);
    for (int k_0 = 0; k_0 < 36; ++k_0)
    {
        *(uint4 *)(A_local + 0) = *(uint4 *)(A + ((k_0 * 256) + (((int)threadIdx.x) * 8)));
        for (int k_2 = 0; k_2 < 8; ++k_2)
        {
            in_thread_C_local[0] = (in_thread_C_local[0] + (A_local[k_2] * ((((((k_2 * 3) & 7) <= 5) ? ((half)((((int)B[(((((((int)blockIdx.x) * 13824) + (((int)threadIdx.y) * 3456)) + (k_0 * 96)) + (((int)threadIdx.x) * 3)) + ((k_2 * 3) >> 3))]) >> ((k_2 * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((int)blockIdx.x) * 13824) + (((int)threadIdx.y) * 3456)) + (k_0 * 96)) + (((int)threadIdx.x) * 3)) + ((k_2 * 3) >> 3))]) >> ((k_2 * 3) & 7)) & ((1 << (8 - ((k_2 * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 13824) + (((int)threadIdx.y) * 3456)) + (k_0 * 96)) + (((int)threadIdx.x) * 3)) + ((k_2 * 3) >> 3)) + 1)]) << (8 - ((k_2 * 3) & 7))) & (7 << (8 - ((k_2 * 3) & 7)))) & 7))))) * Scales[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))]) - Zeros[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))])));
        }
    }
    uint mask[1];
    half t0[1];
    red_buf0[0] = in_thread_C_local[0];
    mask[0] = (__activemask() & ((uint)(0 << (((int)threadIdx.y) * 32))));
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], (((int)threadIdx.y) * 32), 32);
    C[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))] = red_buf0[0];
}

extern "C" __global__ void __launch_bounds__(128) tir_halfxint3_simt_bn4_k36864(half *__restrict__ A, signed char *__restrict__ B, half *__restrict__ Scales, half *__restrict__ Zeros, half *__restrict__ C)
{
    half in_thread_C_local[1];
    half A_local[8];
    half red_buf0[1];
    in_thread_C_local[0] = __float2half_rn(0.000000e+00f);
    for (int k_0 = 0; k_0 < 144; ++k_0)
    {
        *(uint4 *)(A_local + 0) = *(uint4 *)(A + ((k_0 * 256) + (((int)threadIdx.x) * 8)));
        for (int k_2 = 0; k_2 < 8; ++k_2)
        {
            in_thread_C_local[0] = (in_thread_C_local[0] + (A_local[k_2] * ((((((k_2 * 3) & 7) <= 5) ? ((half)((((int)B[(((((((int)blockIdx.x) * 55296) + (((int)threadIdx.y) * 13824)) + (k_0 * 96)) + (((int)threadIdx.x) * 3)) + ((k_2 * 3) >> 3))]) >> ((k_2 * 3) & 7)) & 7)) : ((half)(((signed char)((((int)B[(((((((int)blockIdx.x) * 55296) + (((int)threadIdx.y) * 13824)) + (k_0 * 96)) + (((int)threadIdx.x) * 3)) + ((k_2 * 3) >> 3))]) >> ((k_2 * 3) & 7)) & ((1 << (8 - ((k_2 * 3) & 7))) - 1))) | ((signed char)(((((int)B[((((((((int)blockIdx.x) * 55296) + (((int)threadIdx.y) * 13824)) + (k_0 * 96)) + (((int)threadIdx.x) * 3)) + ((k_2 * 3) >> 3)) + 1)]) << (8 - ((k_2 * 3) & 7))) & (7 << (8 - ((k_2 * 3) & 7)))) & 7))))) * Scales[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))]) - Zeros[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))])));
        }
    }
    uint mask[1];
    half t0[1];
    red_buf0[0] = in_thread_C_local[0];
    mask[0] = (__activemask() & ((uint)(0 << (((int)threadIdx.y) * 32))));
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
    red_buf0[0] = (red_buf0[0] + t0[0]);
    red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], (((int)threadIdx.y) * 32), 32);
    C[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))] = red_buf0[0];
}

