
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

            static inline __device__ __host__ unsigned
            __pack_half2(const half x, const half y) {
            unsigned v0 = *((unsigned short *)&x);
            unsigned v1 = *((unsigned short *)&y);
            return (v1 << 16) | v0;
        }extern "C" __global__ void __launch_bounds__(512) tir_halfxint3_tensorop_64x128x64x1_t0_y2z8_K6656_align8(half* __restrict__ A, signed char* __restrict__ B, half* __restrict__ Scales, half* __restrict__ Zeros, half* __restrict__ C, int m) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[2];
  __shared__ half APad_shared[4608];
  __shared__ half B_rescale_shared[9216];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> APad_shared_wmma_matrix_a[2];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
  for (int i_0_2_init = 0; i_0_2_init < 2; ++i_0_2_init) {
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[i_0_2_init], 0.000000e+00f);
  }
  for (int k_0_0 = 0; k_0_0 < 104; ++k_0_0) {
    __syncthreads();
    *(uint4*)(APad_shared + ((((((int)threadIdx.y) * 2304) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8))) = (((((((int)threadIdx.y) * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 3)) < m) ? *(uint4*)(A + (((((((int)threadIdx.y) * 212992) + (((int)threadIdx.z) * 26624)) + ((((int)threadIdx.x) >> 3) * 6656)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8))) : make_uint4(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f))));
    for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {
      uint2 __1;
        uint2 __2;
          uint2 __3;
          int4 __4;
            int4 __5;
              int4 __6;
              int4 __7;
                int4 v_ = make_int4((((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.y) * 212992)) + (((int)threadIdx.z) * 26624)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), (((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.y) * 212992)) + (((int)threadIdx.z) * 26624)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), (((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.y) * 212992)) + (((int)threadIdx.z) * 26624)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), (((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.y) * 212992)) + (((int)threadIdx.z) * 26624)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)));
                int4 __8;
                  int4 v__1 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                  int4 v__2 = make_int4(2, 2, 2, 2);
                  __8.x = (v__1.x%v__2.x);
                  __8.y = (v__1.y%v__2.y);
                  __8.z = (v__1.z%v__2.z);
                  __8.w = (v__1.w%v__2.w);
                int4 __9;
                  int4 v__3 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                  int4 v__4 = make_int4(2, 2, 2, 2);
                  __9.x = (v__3.x/v__4.x);
                  __9.y = (v__3.y/v__4.y);
                  __9.z = (v__3.z/v__4.z);
                  __9.w = (v__3.w/v__4.w);
                int4 __10;
                ushort4 __11;
                  ushort4 __12;
                    ushort4 __13;
                      int4 v__5 = make_int4(2, 2, 2, 2);
                      int4 v__6 = make_int4(0, 0, 0, 0);
                      __13.x = (v__5.x>=v__6.x);
                      __13.y = (v__5.y>=v__6.y);
                      __13.z = (v__5.z>=v__6.z);
                      __13.w = (v__5.w>=v__6.w);
                    ushort4 __14;
                      int4 v__7 = make_int4(0, 0, 0, 0);
                      __14.x = (__8.x>=v__7.x);
                      __14.y = (__8.y>=v__7.y);
                      __14.z = (__8.z>=v__7.z);
                      __14.w = (__8.w>=v__7.w);
                    __12.x = (__13.x&&__14.x);
                    __12.y = (__13.y&&__14.y);
                    __12.z = (__13.z&&__14.z);
                    __12.w = (__13.w&&__14.w);
                  ushort4 __15;
                    ushort4 __16;
                      int4 v__8 = make_int4(2, 2, 2, 2);
                      int4 v__9 = make_int4(0, 0, 0, 0);
                      __16.x = (v__8.x<v__9.x);
                      __16.y = (v__8.y<v__9.y);
                      __16.z = (v__8.z<v__9.z);
                      __16.w = (v__8.w<v__9.w);
                    ushort4 __17;
                      int4 v__10 = make_int4(0, 0, 0, 0);
                      __17.x = (__8.x<=v__10.x);
                      __17.y = (__8.y<=v__10.y);
                      __17.z = (__8.z<=v__10.z);
                      __17.w = (__8.w<=v__10.w);
                    __15.x = (__16.x&&__17.x);
                    __15.y = (__16.y&&__17.y);
                    __15.z = (__16.z&&__17.z);
                    __15.w = (__16.w&&__17.w);
                  __11.x = (__12.x||__15.x);
                  __11.y = (__12.y||__15.y);
                  __11.z = (__12.z||__15.z);
                  __11.w = (__12.w||__15.w);
                int4 __18;
                  int4 v__11 = make_int4(1, 1, 1, 1);
                  __18.x = (__9.x-v__11.x);
                  __18.y = (__9.y-v__11.y);
                  __18.z = (__9.z-v__11.z);
                  __18.w = (__9.w-v__11.w);
                __10.x = (bool(__11.x)?__9.x:__18.x);
                __10.y = (bool(__11.y)?__9.y:__18.y);
                __10.z = (bool(__11.z)?__9.z:__18.z);
                __10.w = (bool(__11.w)?__9.w:__18.w);
                __7.x = (v_.x+__10.x);
                __7.y = (v_.y+__10.y);
                __7.z = (v_.z+__10.z);
                __7.w = (v_.w+__10.w);
              int v__12 = ((0x000000ff << 0) & (B[__7.x] << 0))|((0x000000ff << 8) & (B[__7.y] << 8))|((0x000000ff << 16) & (B[__7.z] << 16))|((0x000000ff << 24) & (B[__7.w] << 24));
              __6.x = (int)(((char)(v__12 >> 0)));
              __6.y = (int)(((char)(v__12 >> 8)));
              __6.z = (int)(((char)(v__12 >> 16)));
              __6.w = (int)(((char)(v__12 >> 24)));
              int4 __19;
                int4 __20;
                  int4 v__13 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                  int4 v__14 = make_int4(2, 2, 2, 2);
                  __20.x = (v__13.x%v__14.x);
                  __20.y = (v__13.y%v__14.y);
                  __20.z = (v__13.z%v__14.z);
                  __20.w = (v__13.w%v__14.w);
                int4 __21;
                ushort4 __22;
                  ushort4 __23;
                    ushort4 __24;
                      int4 v__15 = make_int4(2, 2, 2, 2);
                      int4 v__16 = make_int4(0, 0, 0, 0);
                      __24.x = (v__15.x>=v__16.x);
                      __24.y = (v__15.y>=v__16.y);
                      __24.z = (v__15.z>=v__16.z);
                      __24.w = (v__15.w>=v__16.w);
                    ushort4 __25;
                      int4 v__17 = make_int4(0, 0, 0, 0);
                      __25.x = (__20.x>=v__17.x);
                      __25.y = (__20.y>=v__17.y);
                      __25.z = (__20.z>=v__17.z);
                      __25.w = (__20.w>=v__17.w);
                    __23.x = (__24.x&&__25.x);
                    __23.y = (__24.y&&__25.y);
                    __23.z = (__24.z&&__25.z);
                    __23.w = (__24.w&&__25.w);
                  ushort4 __26;
                    ushort4 __27;
                      int4 v__18 = make_int4(2, 2, 2, 2);
                      int4 v__19 = make_int4(0, 0, 0, 0);
                      __27.x = (v__18.x<v__19.x);
                      __27.y = (v__18.y<v__19.y);
                      __27.z = (v__18.z<v__19.z);
                      __27.w = (v__18.w<v__19.w);
                    ushort4 __28;
                      int4 v__20 = make_int4(0, 0, 0, 0);
                      __28.x = (__20.x<=v__20.x);
                      __28.y = (__20.y<=v__20.y);
                      __28.z = (__20.z<=v__20.z);
                      __28.w = (__20.w<=v__20.w);
                    __26.x = (__27.x&&__28.x);
                    __26.y = (__27.y&&__28.y);
                    __26.z = (__27.z&&__28.z);
                    __26.w = (__27.w&&__28.w);
                  __22.x = (__23.x||__26.x);
                  __22.y = (__23.y||__26.y);
                  __22.z = (__23.z||__26.z);
                  __22.w = (__23.w||__26.w);
                int4 __29;
                  int4 v__21 = make_int4(2, 2, 2, 2);
                  __29.x = (__20.x+v__21.x);
                  __29.y = (__20.y+v__21.y);
                  __29.z = (__20.z+v__21.z);
                  __29.w = (__20.w+v__21.w);
                __21.x = (bool(__22.x)?__20.x:__29.x);
                __21.y = (bool(__22.y)?__20.y:__29.y);
                __21.z = (bool(__22.z)?__20.z:__29.z);
                __21.w = (bool(__22.w)?__20.w:__29.w);
                int4 v__22 = make_int4(4, 4, 4, 4);
                __19.x = (__21.x*v__22.x);
                __19.y = (__21.y*v__22.y);
                __19.z = (__21.z*v__22.z);
                __19.w = (__21.w*v__22.w);
              __5.x = (__6.x >> __19.x);
              __5.y = (__6.y >> __19.y);
              __5.z = (__6.z >> __19.z);
              __5.w = (__6.w >> __19.w);
            int4 v__23 = make_int4(15, 15, 15, 15);
            __4.x = (__5.x & v__23.x);
            __4.y = (__5.y & v__23.y);
            __4.z = (__5.z & v__23.z);
            __4.w = (__5.w & v__23.w);
          ((half2*)(&(__3.x)))->x = (half)(__4.x);
          ((half2*)(&(__3.x)))->y = (half)(__4.y);
          ((half2*)(&(__3.y)))->x = (half)(__4.z);
          ((half2*)(&(__3.y)))->y = (half)(__4.w);
          uint2 v__24 = make_uint2(__pack_half2(Scales[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Scales[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]), __pack_half2(Scales[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Scales[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]));
          ((half2*)(&(__2.x)))->x = (((half2*)(&(__3.x)))->x*((half2*)(&(v__24.x)))->x);
          ((half2*)(&(__2.x)))->y = (((half2*)(&(__3.x)))->y*((half2*)(&(v__24.x)))->y);
          ((half2*)(&(__2.y)))->x = (((half2*)(&(__3.y)))->x*((half2*)(&(v__24.y)))->x);
          ((half2*)(&(__2.y)))->y = (((half2*)(&(__3.y)))->y*((half2*)(&(v__24.y)))->y);
        uint2 v__25 = make_uint2(__pack_half2(Zeros[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Zeros[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]), __pack_half2(Zeros[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Zeros[(((((((int)blockIdx.x) * 128) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.z) * 8)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]));
        ((half2*)(&(__1.x)))->x = (((half2*)(&(__2.x)))->x-((half2*)(&(v__25.x)))->x);
        ((half2*)(&(__1.x)))->y = (((half2*)(&(__2.x)))->y-((half2*)(&(v__25.x)))->y);
        ((half2*)(&(__1.y)))->x = (((half2*)(&(__2.y)))->x-((half2*)(&(v__25.y)))->x);
        ((half2*)(&(__1.y)))->y = (((half2*)(&(__2.y)))->y-((half2*)(&(v__25.y)))->y);
      *(uint2*)(B_rescale_shared + (((((((int)threadIdx.y) * 4608) + (((int)threadIdx.z) * 576)) + (ax0_ax1_fused_2 * 144)) + ((((int)threadIdx.x) >> 4) * 72)) + ((((int)threadIdx.x) & 15) * 4))) = __1;
    }
    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(APad_shared_wmma_matrix_a[ax0_0], (&(APad_shared[(((((int)threadIdx.y) * 2304) + (ax0_0 * 1152)) + (k_0_1 * 16))])), 72);
      }
      nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((int)threadIdx.z) * 1152) + (k_0_1 * 16))])), 72);
      for (int i_0_2 = 0; i_0_2 < 2; ++i_0_2) {
        nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2], APad_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2]);
      }
    }
  }
  for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1) {
    nvcuda::wmma::store_matrix_sync((&(C[((((((int)threadIdx.y) * 573440) + (ax0_0_1 * 286720)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[ax0_0_1], 17920, nvcuda::wmma::mem_row_major);
  }
}

