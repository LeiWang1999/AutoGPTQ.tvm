
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
extern "C" __global__ void __launch_bounds__(128) tir_halfxint4_simt_bn4_k6656(half* __restrict__ A, signed char* __restrict__ B, half* __restrict__ Scales, half* __restrict__ Zeros, half* __restrict__ C) {
  half in_thread_C_local[1];
  half A_local[8];
  half red_buf0[1];
  in_thread_C_local[0] = __float2half_rn(0.000000e+00f);
  for (int k_0 = 0; k_0 < 26; ++k_0) {
    *(uint4*)(A_local + 0) = *(uint4*)(A + ((k_0 * 256) + (((int)threadIdx.x) * 8)));
    for (int k_2 = 0; k_2 < 8; ++k_2) {
      in_thread_C_local[0] = (in_thread_C_local[0] + (A_local[k_2] * ((((half)((((int)B[(((((((int)blockIdx.x) * 13312) + (((int)threadIdx.y) * 3328)) + (k_0 * 128)) + (((int)threadIdx.x) * 4)) + (k_2 >> 1))]) >> ((k_2 & 1) * 4)) & 15)) * Scales[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))]) - Zeros[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))])));
    }
  }
  uint mask[1];
  half t0[1];
  red_buf0[0] = in_thread_C_local[0];
  mask[0] = (__activemask() & ((uint)(0 << (((int)threadIdx.y) * 32))));
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], (((int)threadIdx.y) * 32), 32);
  C[((((int)blockIdx.x) * 4) + ((int)threadIdx.y))] = red_buf0[0];
}

