
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

            static inline __device__ __host__ unsigned
            __pack_half2(const half x, const half y) {
            unsigned v0 = *((unsigned short *)&x);
            unsigned v1 = *((unsigned short *)&y);
            return (v1 << 16) | v0;
        }extern "C" __global__ void __launch_bounds__(256) tir_halfxint3_tensorop_32x128x64x2_t0_y1z8_K6656_align8(half* __restrict__ A, signed char* __restrict__ B, half* __restrict__ Scales, half* __restrict__ Zeros, half* __restrict__ C, int m) {
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> C_wmma_accumulator[2];
  __shared__ half APad_shared[4608];
  __shared__ half B_rescale_shared[18432];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> APad_shared_wmma_matrix_a[2];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> B_rescale_shared_wmma_matrix_b[1];
  for (int i_0_2_init = 0; i_0_2_init < 2; ++i_0_2_init) {
    nvcuda::wmma::fill_fragment(C_wmma_accumulator[i_0_2_init], 0.000000e+00f);
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }"
      : "=r"(addr)
      : "l"((void *)(APad_shared + (((((int)threadIdx.z) * 288) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8))))
    );
    int pred_guard = (int)(((((int)threadIdx.z) * 4) + (((int)threadIdx.x) >> 3)) < m);
    __asm__ __volatile__(
        "{  .reg .pred p;"
        "  setp.ne.b32 p, %0, 0;"
      #if TVM_ENABLE_L2_PREFETCH
        " @p cp.async.cg.shared.global.L2::128B [%1], [%2], %3;"
      #else
        " @p cp.async.cg.shared.global [%1], [%2], %3;"
      #endif
      "  @!p st.shared.v4.u32 [%1], {%4, %5, %6, %7};}"
        :: "r"(pred_guard), "r"(addr), "l"((void*)(A + (((((int)threadIdx.z) * 26624) + ((((int)threadIdx.x) >> 3) * 6656)) + ((((int)threadIdx.x) & 7) * 8)))), "n"(16), "r"(0), "r"(0), "r"(0),"r"(0)
    );
  }
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 8; ++ax0_ax1_fused_2) {
    uint2 __1;
      uint2 __2;
        uint2 __3;
        int4 __4;
          int4 __5;
            int4 __6;
            int4 __7;
              int4 v_ = make_int4((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + ((((int)threadIdx.x) & 15) * 2)), (((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + ((((int)threadIdx.x) & 15) * 2)), (((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + ((((int)threadIdx.x) & 15) * 2)), (((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + ((((int)threadIdx.x) & 15) * 2)));
              int4 __8;
                int4 v__1 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                int4 v__2 = make_int4(2, 2, 2, 2);
                __8.x = (v__1.x%v__2.x);
                __8.y = (v__1.y%v__2.y);
                __8.z = (v__1.z%v__2.z);
                __8.w = (v__1.w%v__2.w);
              int4 __9;
                int4 v__3 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                int4 v__4 = make_int4(2, 2, 2, 2);
                __9.x = (v__3.x/v__4.x);
                __9.y = (v__3.y/v__4.y);
                __9.z = (v__3.z/v__4.z);
                __9.w = (v__3.w/v__4.w);
              int4 __10;
              ushort4 __11;
                ushort4 __12;
                  ushort4 __13;
                    int4 v__5 = make_int4(2, 2, 2, 2);
                    int4 v__6 = make_int4(0, 0, 0, 0);
                    __13.x = (v__5.x>=v__6.x);
                    __13.y = (v__5.y>=v__6.y);
                    __13.z = (v__5.z>=v__6.z);
                    __13.w = (v__5.w>=v__6.w);
                  ushort4 __14;
                    int4 v__7 = make_int4(0, 0, 0, 0);
                    __14.x = (__8.x>=v__7.x);
                    __14.y = (__8.y>=v__7.y);
                    __14.z = (__8.z>=v__7.z);
                    __14.w = (__8.w>=v__7.w);
                  __12.x = (__13.x&&__14.x);
                  __12.y = (__13.y&&__14.y);
                  __12.z = (__13.z&&__14.z);
                  __12.w = (__13.w&&__14.w);
                ushort4 __15;
                  ushort4 __16;
                    int4 v__8 = make_int4(2, 2, 2, 2);
                    int4 v__9 = make_int4(0, 0, 0, 0);
                    __16.x = (v__8.x<v__9.x);
                    __16.y = (v__8.y<v__9.y);
                    __16.z = (v__8.z<v__9.z);
                    __16.w = (v__8.w<v__9.w);
                  ushort4 __17;
                    int4 v__10 = make_int4(0, 0, 0, 0);
                    __17.x = (__8.x<=v__10.x);
                    __17.y = (__8.y<=v__10.y);
                    __17.z = (__8.z<=v__10.z);
                    __17.w = (__8.w<=v__10.w);
                  __15.x = (__16.x&&__17.x);
                  __15.y = (__16.y&&__17.y);
                  __15.z = (__16.z&&__17.z);
                  __15.w = (__16.w&&__17.w);
                __11.x = (__12.x||__15.x);
                __11.y = (__12.y||__15.y);
                __11.z = (__12.z||__15.z);
                __11.w = (__12.w||__15.w);
              int4 __18;
                int4 v__11 = make_int4(1, 1, 1, 1);
                __18.x = (__9.x-v__11.x);
                __18.y = (__9.y-v__11.y);
                __18.z = (__9.z-v__11.z);
                __18.w = (__9.w-v__11.w);
              __10.x = (bool(__11.x)?__9.x:__18.x);
              __10.y = (bool(__11.y)?__9.y:__18.y);
              __10.z = (bool(__11.z)?__9.z:__18.z);
              __10.w = (bool(__11.w)?__9.w:__18.w);
              __7.x = (v_.x+__10.x);
              __7.y = (v_.y+__10.y);
              __7.z = (v_.z+__10.z);
              __7.w = (v_.w+__10.w);
            int v__12 = ((0x000000ff << 0) & (B[__7.x] << 0))|((0x000000ff << 8) & (B[__7.y] << 8))|((0x000000ff << 16) & (B[__7.z] << 16))|((0x000000ff << 24) & (B[__7.w] << 24));
            __6.x = (int)(((char)(v__12 >> 0)));
            __6.y = (int)(((char)(v__12 >> 8)));
            __6.z = (int)(((char)(v__12 >> 16)));
            __6.w = (int)(((char)(v__12 >> 24)));
            int4 __19;
              int4 __20;
                int4 v__13 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                int4 v__14 = make_int4(2, 2, 2, 2);
                __20.x = (v__13.x%v__14.x);
                __20.y = (v__13.y%v__14.y);
                __20.z = (v__13.z%v__14.z);
                __20.w = (v__13.w%v__14.w);
              int4 __21;
              ushort4 __22;
                ushort4 __23;
                  ushort4 __24;
                    int4 v__15 = make_int4(2, 2, 2, 2);
                    int4 v__16 = make_int4(0, 0, 0, 0);
                    __24.x = (v__15.x>=v__16.x);
                    __24.y = (v__15.y>=v__16.y);
                    __24.z = (v__15.z>=v__16.z);
                    __24.w = (v__15.w>=v__16.w);
                  ushort4 __25;
                    int4 v__17 = make_int4(0, 0, 0, 0);
                    __25.x = (__20.x>=v__17.x);
                    __25.y = (__20.y>=v__17.y);
                    __25.z = (__20.z>=v__17.z);
                    __25.w = (__20.w>=v__17.w);
                  __23.x = (__24.x&&__25.x);
                  __23.y = (__24.y&&__25.y);
                  __23.z = (__24.z&&__25.z);
                  __23.w = (__24.w&&__25.w);
                ushort4 __26;
                  ushort4 __27;
                    int4 v__18 = make_int4(2, 2, 2, 2);
                    int4 v__19 = make_int4(0, 0, 0, 0);
                    __27.x = (v__18.x<v__19.x);
                    __27.y = (v__18.y<v__19.y);
                    __27.z = (v__18.z<v__19.z);
                    __27.w = (v__18.w<v__19.w);
                  ushort4 __28;
                    int4 v__20 = make_int4(0, 0, 0, 0);
                    __28.x = (__20.x<=v__20.x);
                    __28.y = (__20.y<=v__20.y);
                    __28.z = (__20.z<=v__20.z);
                    __28.w = (__20.w<=v__20.w);
                  __26.x = (__27.x&&__28.x);
                  __26.y = (__27.y&&__28.y);
                  __26.z = (__27.z&&__28.z);
                  __26.w = (__27.w&&__28.w);
                __22.x = (__23.x||__26.x);
                __22.y = (__23.y||__26.y);
                __22.z = (__23.z||__26.z);
                __22.w = (__23.w||__26.w);
              int4 __29;
                int4 v__21 = make_int4(2, 2, 2, 2);
                __29.x = (__20.x+v__21.x);
                __29.y = (__20.y+v__21.y);
                __29.z = (__20.z+v__21.z);
                __29.w = (__20.w+v__21.w);
              __21.x = (bool(__22.x)?__20.x:__29.x);
              __21.y = (bool(__22.y)?__20.y:__29.y);
              __21.z = (bool(__22.z)?__20.z:__29.z);
              __21.w = (bool(__22.w)?__20.w:__29.w);
              int4 v__22 = make_int4(4, 4, 4, 4);
              __19.x = (__21.x*v__22.x);
              __19.y = (__21.y*v__22.y);
              __19.z = (__21.z*v__22.z);
              __19.w = (__21.w*v__22.w);
            __5.x = (__6.x >> __19.x);
            __5.y = (__6.y >> __19.y);
            __5.z = (__6.z >> __19.z);
            __5.w = (__6.w >> __19.w);
          int4 v__23 = make_int4(15, 15, 15, 15);
          __4.x = (__5.x & v__23.x);
          __4.y = (__5.y & v__23.y);
          __4.z = (__5.z & v__23.z);
          __4.w = (__5.w & v__23.w);
        ((half2*)(&(__3.x)))->x = (half)(__4.x);
        ((half2*)(&(__3.x)))->y = (half)(__4.y);
        ((half2*)(&(__3.y)))->x = (half)(__4.z);
        ((half2*)(&(__3.y)))->y = (half)(__4.w);
        uint2 v__24 = make_uint2(__pack_half2(Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]), __pack_half2(Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]));
        ((half2*)(&(__2.x)))->x = (((half2*)(&(__3.x)))->x*((half2*)(&(v__24.x)))->x);
        ((half2*)(&(__2.x)))->y = (((half2*)(&(__3.x)))->y*((half2*)(&(v__24.x)))->y);
        ((half2*)(&(__2.y)))->x = (((half2*)(&(__3.y)))->x*((half2*)(&(v__24.y)))->x);
        ((half2*)(&(__2.y)))->y = (((half2*)(&(__3.y)))->y*((half2*)(&(v__24.y)))->y);
      uint2 v__25 = make_uint2(__pack_half2(Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]), __pack_half2(Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))], Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2 * 2)) + (((int)threadIdx.x) >> 4))]));
      ((half2*)(&(__1.x)))->x = (((half2*)(&(__2.x)))->x-((half2*)(&(v__25.x)))->x);
      ((half2*)(&(__1.x)))->y = (((half2*)(&(__2.x)))->y-((half2*)(&(v__25.x)))->y);
      ((half2*)(&(__1.y)))->x = (((half2*)(&(__2.y)))->x-((half2*)(&(v__25.y)))->x);
      ((half2*)(&(__1.y)))->y = (((half2*)(&(__2.y)))->y-((half2*)(&(v__25.y)))->y);
    *(uint2*)(B_rescale_shared + ((((((int)threadIdx.z) * 1152) + (ax0_ax1_fused_2 * 144)) + ((((int)threadIdx.x) >> 4) * 72)) + ((((int)threadIdx.x) & 15) * 4))) = __1;
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_0 = 0; k_0_0 < 103; ++k_0_0) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }"
      : "=r"(addr)
      : "l"((void *)(APad_shared + ((((((k_0_0 + 1) & 1) * 2304) + (((int)threadIdx.z) * 288)) + ((((int)threadIdx.x) >> 3) * 72)) + ((((int)threadIdx.x) & 7) * 8))))
    );
    int pred_guard = (int)(((((int)threadIdx.z) * 4) + (((int)threadIdx.x) >> 3)) < m);
    __asm__ __volatile__(
        "{  .reg .pred p;"
        "  setp.ne.b32 p, %0, 0;"
      #if TVM_ENABLE_L2_PREFETCH
        " @p cp.async.cg.shared.global.L2::128B [%1], [%2], %3;"
      #else
        " @p cp.async.cg.shared.global [%1], [%2], %3;"
      #endif
      "  @!p st.shared.v4.u32 [%1], {%4, %5, %6, %7};}"
        :: "r"(pred_guard), "r"(addr), "l"((void*)(A + (((((((int)threadIdx.z) * 26624) + ((((int)threadIdx.x) >> 3) * 6656)) + (k_0_0 * 64)) + ((((int)threadIdx.x) & 7) * 8)) + 64))), "n"(16), "r"(0), "r"(0), "r"(0),"r"(0)
    );
  }
    for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 8; ++ax0_ax1_fused_2_1) {
      uint2 __30;
        uint2 __31;
          uint2 __32;
          int4 __33;
            int4 __34;
              int4 __35;
              int4 __36;
                int4 __37;
                  int4 v__26 = make_int4(((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2_1 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), ((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2_1 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), ((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2_1 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)), ((((((((int)blockIdx.x) * 425984) + (((int)threadIdx.z) * 53248)) + (ax0_ax1_fused_2_1 * 6656)) + ((((int)threadIdx.x) >> 4) * 3328)) + (k_0_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)));
                  int4 __38;
                    int4 v__27 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                    int4 v__28 = make_int4(2, 2, 2, 2);
                    __38.x = (v__27.x%v__28.x);
                    __38.y = (v__27.y%v__28.y);
                    __38.z = (v__27.z%v__28.z);
                    __38.w = (v__27.w%v__28.w);
                  int4 __39;
                    int4 v__29 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                    int4 v__30 = make_int4(2, 2, 2, 2);
                    __39.x = (v__29.x/v__30.x);
                    __39.y = (v__29.y/v__30.y);
                    __39.z = (v__29.z/v__30.z);
                    __39.w = (v__29.w/v__30.w);
                  int4 __40;
                  ushort4 __41;
                    ushort4 __42;
                      ushort4 __43;
                        int4 v__31 = make_int4(2, 2, 2, 2);
                        int4 v__32 = make_int4(0, 0, 0, 0);
                        __43.x = (v__31.x>=v__32.x);
                        __43.y = (v__31.y>=v__32.y);
                        __43.z = (v__31.z>=v__32.z);
                        __43.w = (v__31.w>=v__32.w);
                      ushort4 __44;
                        int4 v__33 = make_int4(0, 0, 0, 0);
                        __44.x = (__38.x>=v__33.x);
                        __44.y = (__38.y>=v__33.y);
                        __44.z = (__38.z>=v__33.z);
                        __44.w = (__38.w>=v__33.w);
                      __42.x = (__43.x&&__44.x);
                      __42.y = (__43.y&&__44.y);
                      __42.z = (__43.z&&__44.z);
                      __42.w = (__43.w&&__44.w);
                    ushort4 __45;
                      ushort4 __46;
                        int4 v__34 = make_int4(2, 2, 2, 2);
                        int4 v__35 = make_int4(0, 0, 0, 0);
                        __46.x = (v__34.x<v__35.x);
                        __46.y = (v__34.y<v__35.y);
                        __46.z = (v__34.z<v__35.z);
                        __46.w = (v__34.w<v__35.w);
                      ushort4 __47;
                        int4 v__36 = make_int4(0, 0, 0, 0);
                        __47.x = (__38.x<=v__36.x);
                        __47.y = (__38.y<=v__36.y);
                        __47.z = (__38.z<=v__36.z);
                        __47.w = (__38.w<=v__36.w);
                      __45.x = (__46.x&&__47.x);
                      __45.y = (__46.y&&__47.y);
                      __45.z = (__46.z&&__47.z);
                      __45.w = (__46.w&&__47.w);
                    __41.x = (__42.x||__45.x);
                    __41.y = (__42.y||__45.y);
                    __41.z = (__42.z||__45.z);
                    __41.w = (__42.w||__45.w);
                  int4 __48;
                    int4 v__37 = make_int4(1, 1, 1, 1);
                    __48.x = (__39.x-v__37.x);
                    __48.y = (__39.y-v__37.y);
                    __48.z = (__39.z-v__37.z);
                    __48.w = (__39.w-v__37.w);
                  __40.x = (bool(__41.x)?__39.x:__48.x);
                  __40.y = (bool(__41.y)?__39.y:__48.y);
                  __40.z = (bool(__41.z)?__39.z:__48.z);
                  __40.w = (bool(__41.w)?__39.w:__48.w);
                  __37.x = (v__26.x+__40.x);
                  __37.y = (v__26.y+__40.y);
                  __37.z = (v__26.z+__40.z);
                  __37.w = (v__26.w+__40.w);
                int4 v__38 = make_int4(32, 32, 32, 32);
                __36.x = (__37.x+v__38.x);
                __36.y = (__37.y+v__38.y);
                __36.z = (__37.z+v__38.z);
                __36.w = (__37.w+v__38.w);
              int v__39 = ((0x000000ff << 0) & (B[__36.x] << 0))|((0x000000ff << 8) & (B[__36.y] << 8))|((0x000000ff << 16) & (B[__36.z] << 16))|((0x000000ff << 24) & (B[__36.w] << 24));
              __35.x = (int)(((char)(v__39 >> 0)));
              __35.y = (int)(((char)(v__39 >> 8)));
              __35.z = (int)(((char)(v__39 >> 16)));
              __35.w = (int)(((char)(v__39 >> 24)));
              int4 __49;
                int4 __50;
                  int4 v__40 = make_int4((0)+(1*0), (0)+(1*1), (0)+(1*2), (0)+(1*3));
                  int4 v__41 = make_int4(2, 2, 2, 2);
                  __50.x = (v__40.x%v__41.x);
                  __50.y = (v__40.y%v__41.y);
                  __50.z = (v__40.z%v__41.z);
                  __50.w = (v__40.w%v__41.w);
                int4 __51;
                ushort4 __52;
                  ushort4 __53;
                    ushort4 __54;
                      int4 v__42 = make_int4(2, 2, 2, 2);
                      int4 v__43 = make_int4(0, 0, 0, 0);
                      __54.x = (v__42.x>=v__43.x);
                      __54.y = (v__42.y>=v__43.y);
                      __54.z = (v__42.z>=v__43.z);
                      __54.w = (v__42.w>=v__43.w);
                    ushort4 __55;
                      int4 v__44 = make_int4(0, 0, 0, 0);
                      __55.x = (__50.x>=v__44.x);
                      __55.y = (__50.y>=v__44.y);
                      __55.z = (__50.z>=v__44.z);
                      __55.w = (__50.w>=v__44.w);
                    __53.x = (__54.x&&__55.x);
                    __53.y = (__54.y&&__55.y);
                    __53.z = (__54.z&&__55.z);
                    __53.w = (__54.w&&__55.w);
                  ushort4 __56;
                    ushort4 __57;
                      int4 v__45 = make_int4(2, 2, 2, 2);
                      int4 v__46 = make_int4(0, 0, 0, 0);
                      __57.x = (v__45.x<v__46.x);
                      __57.y = (v__45.y<v__46.y);
                      __57.z = (v__45.z<v__46.z);
                      __57.w = (v__45.w<v__46.w);
                    ushort4 __58;
                      int4 v__47 = make_int4(0, 0, 0, 0);
                      __58.x = (__50.x<=v__47.x);
                      __58.y = (__50.y<=v__47.y);
                      __58.z = (__50.z<=v__47.z);
                      __58.w = (__50.w<=v__47.w);
                    __56.x = (__57.x&&__58.x);
                    __56.y = (__57.y&&__58.y);
                    __56.z = (__57.z&&__58.z);
                    __56.w = (__57.w&&__58.w);
                  __52.x = (__53.x||__56.x);
                  __52.y = (__53.y||__56.y);
                  __52.z = (__53.z||__56.z);
                  __52.w = (__53.w||__56.w);
                int4 __59;
                  int4 v__48 = make_int4(2, 2, 2, 2);
                  __59.x = (__50.x+v__48.x);
                  __59.y = (__50.y+v__48.y);
                  __59.z = (__50.z+v__48.z);
                  __59.w = (__50.w+v__48.w);
                __51.x = (bool(__52.x)?__50.x:__59.x);
                __51.y = (bool(__52.y)?__50.y:__59.y);
                __51.z = (bool(__52.z)?__50.z:__59.z);
                __51.w = (bool(__52.w)?__50.w:__59.w);
                int4 v__49 = make_int4(4, 4, 4, 4);
                __49.x = (__51.x*v__49.x);
                __49.y = (__51.y*v__49.y);
                __49.z = (__51.z*v__49.z);
                __49.w = (__51.w*v__49.w);
              __34.x = (__35.x >> __49.x);
              __34.y = (__35.y >> __49.y);
              __34.z = (__35.z >> __49.z);
              __34.w = (__35.w >> __49.w);
            int4 v__50 = make_int4(15, 15, 15, 15);
            __33.x = (__34.x & v__50.x);
            __33.y = (__34.y & v__50.y);
            __33.z = (__34.z & v__50.z);
            __33.w = (__34.w & v__50.w);
          ((half2*)(&(__32.x)))->x = (half)(__33.x);
          ((half2*)(&(__32.x)))->y = (half)(__33.y);
          ((half2*)(&(__32.y)))->x = (half)(__33.z);
          ((half2*)(&(__32.y)))->y = (half)(__33.w);
          uint2 v__51 = make_uint2(__pack_half2(Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))], Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))]), __pack_half2(Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))], Scales[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))]));
          ((half2*)(&(__31.x)))->x = (((half2*)(&(__32.x)))->x*((half2*)(&(v__51.x)))->x);
          ((half2*)(&(__31.x)))->y = (((half2*)(&(__32.x)))->y*((half2*)(&(v__51.x)))->y);
          ((half2*)(&(__31.y)))->x = (((half2*)(&(__32.y)))->x*((half2*)(&(v__51.y)))->x);
          ((half2*)(&(__31.y)))->y = (((half2*)(&(__32.y)))->y*((half2*)(&(v__51.y)))->y);
        uint2 v__52 = make_uint2(__pack_half2(Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))], Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))]), __pack_half2(Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))], Zeros[((((((int)blockIdx.x) * 128) + (((int)threadIdx.z) * 16)) + (ax0_ax1_fused_2_1 * 2)) + (((int)threadIdx.x) >> 4))]));
        ((half2*)(&(__30.x)))->x = (((half2*)(&(__31.x)))->x-((half2*)(&(v__52.x)))->x);
        ((half2*)(&(__30.x)))->y = (((half2*)(&(__31.x)))->y-((half2*)(&(v__52.x)))->y);
        ((half2*)(&(__30.y)))->x = (((half2*)(&(__31.y)))->x-((half2*)(&(v__52.y)))->x);
        ((half2*)(&(__30.y)))->y = (((half2*)(&(__31.y)))->y-((half2*)(&(v__52.y)))->y);
      *(uint2*)(B_rescale_shared + (((((((k_0_0 + 1) & 1) * 9216) + (((int)threadIdx.z) * 1152)) + (ax0_ax1_fused_2_1 * 144)) + ((((int)threadIdx.x) >> 4) * 72)) + ((((int)threadIdx.x) & 15) * 4))) = __30;
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1) {
      for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
        nvcuda::wmma::load_matrix_sync(APad_shared_wmma_matrix_a[ax0_0], (&(APad_shared[((((k_0_0 & 1) * 2304) + (ax0_0 * 1152)) + (k_0_1 * 16))])), 72);
      }
      nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[((((k_0_0 & 1) * 9216) + (((int)threadIdx.z) * 1152)) + (k_0_1 * 16))])), 72);
      for (int i_0_2 = 0; i_0_2 < 2; ++i_0_2) {
        nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2], APad_shared_wmma_matrix_a[i_0_2], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2]);
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int k_0_1_1 = 0; k_0_1_1 < 4; ++k_0_1_1) {
    for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1) {
      nvcuda::wmma::load_matrix_sync(APad_shared_wmma_matrix_a[ax0_0_1], (&(APad_shared[(((ax0_0_1 * 1152) + (k_0_1_1 * 16)) + 2304)])), 72);
    }
    nvcuda::wmma::load_matrix_sync(B_rescale_shared_wmma_matrix_b[0], (&(B_rescale_shared[(((((int)threadIdx.z) * 1152) + (k_0_1_1 * 16)) + 9216)])), 72);
    for (int i_0_2_1 = 0; i_0_2_1 < 2; ++i_0_2_1) {
      nvcuda::wmma::mma_sync(C_wmma_accumulator[i_0_2_1], APad_shared_wmma_matrix_a[i_0_2_1], B_rescale_shared_wmma_matrix_b[0], C_wmma_accumulator[i_0_2_1]);
    }
  }
  for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2) {
    nvcuda::wmma::store_matrix_sync((&(C[(((ax0_0_2 * 286720) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.z) * 16))])), C_wmma_accumulator[ax0_0_2], 17920, nvcuda::wmma::mem_row_major);
  }
}

