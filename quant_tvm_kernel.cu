#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "./src/kernels/gemm_kernel.cu"
#include "./src/kernels/gemv_kernel.cu"

void quant_kernel_3b_cuda(
    torch::Tensor mat_A,
    torch::Tensor mat_B,
    torch::Tensor mat_C,
    torch::Tensor scales,
    torch::Tensor zeros)
{
    int M = 1;
    for (int i = 0; i < mat_A.dim() - 1; i++)
        M *= mat_A.size(i);
    int N = mat_B.size(0);
    int K = mat_A.size(mat_A.dim() - 1);
    int K_Compressed = mat_B.size(1);

    if (M == 1)
    {
            const int num_warps = 4;
            const int warp_size = 32;
            dim3 blocks(
                (N + num_warps - 1) / num_warps, 1, 1);
            dim3 threads(warp_size, num_warps, 1);
            if (K == 9216)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                mat_A.type(), "tir_halfxint3_simt_bn4_k9216", ([&]
                                                               { tir_halfxint3_simt_bn4_k9216<<<blocks, threads>>>(
                                                                     (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                     (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
        
            else if (K == 36864)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_simt_bn4_k36864", ([&]
                                                                { tir_halfxint3_simt_bn4_k36864<<<blocks, threads>>>(
                                                                        (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                        (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
    } else if (M == 16){
            const int warp_size = 32;
            const int warp_row = 1;
            const int warp_col = 4;
            const int raster = 0;
            const int BM = 16;
            const int BN = 64;
            const int BK = 64;
            dim3 blocks(
                (N + BN - 1) / BN, (M + BM - 1) / BM, 1);
            dim3 threads(warp_size, warp_row, warp_col);
            if (K == 9216)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_16x64x64x3_t0_y1z4_K9216_align8", ([&]
                                                                                             { tir_halfxint3_tensorop_16x64x64x3_t0_y1z4_K9216_align8<<<blocks, threads>>>(
                                                                                                   (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                   (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
            else if (K == 36864)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_16x64x64x3_t0_y1z4_K36864_align8", ([&]
                                                                                              { tir_halfxint3_tensorop_16x64x64x3_t0_y1z4_K36864_align8<<<blocks, threads>>>(
                                                                                                    (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                    (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
    } else if (M == 32){
            const int warp_size = 32;
            const int warp_row = 2;
            const int warp_col = 4;
            const int raster = 0;
            const int BM = 32;
            const int BN = 64;
            const int BK = 64;
            dim3 blocks(
                (N + BN - 1) / BN, (M + BM - 1) / BM, 1);
            dim3 threads(warp_size, warp_row, warp_col);
            if (K == 9216)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_32x64x64x2_t0_y2z4_K9216_align8", ([&]
                                                                                             { tir_halfxint3_tensorop_32x64x64x2_t0_y2z4_K9216_align8<<<blocks, threads>>>(
                                                                                                   (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                   (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
            else if (K == 36864)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_32x64x64x2_t0_y2z4_K36864_align8", ([&]
                                                                                              { tir_halfxint3_tensorop_32x64x64x2_t0_y2z4_K36864_align8<<<blocks, threads>>>(
                                                                                                    (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                    (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
    } else if (M == 64){
            const int warp_size = 32;
            const int warp_row = 2;
            const int warp_col = 4;
            const int raster = 0;
            const int BM = 64;
            const int BN = 64;
            const int BK = 64;
            dim3 blocks(
                (N + BN - 1) / BN, (M + BM - 1) / BM, 1);
            dim3 threads(warp_size, warp_row, warp_col);
            
            if (K == 9216)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_64x64x64x2_t0_y2z4_K9216_align8", ([&]
                                                                                             { tir_halfxint3_tensorop_64x64x64x2_t0_y2z4_K9216_align8<<<blocks, threads>>>(
                                                                                                   (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                   (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
            else if (K == 36864)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_64x64x64x2_t0_y2z4_K36864_align8", ([&]
                                                                                              { tir_halfxint3_tensorop_64x64x64x2_t0_y2z4_K36864_align8<<<blocks, threads>>>(
                                                                                                    (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                    (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
    } else if (M == 128){
            const int warp_size = 32;
            const int warp_row = 1;
            const int warp_col = 2;
            const int raster = 0;
            const int BM = 64;
            const int BN = 128;
            const int BK = 32;
            dim3 blocks(
                (N + BN - 1) / BN, (M + BM - 1) / BM, 1);
            dim3 threads(warp_size, warp_row, warp_col);
            if (K == 9216)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_64x128x32x3_t0_y1z2_K9216_align8", ([&]
                                                                                              { tir_halfxint3_tensorop_64x128x32x3_t0_y1z2_K9216_align8<<<blocks, threads>>>(
                                                                                                    (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                    (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
            else if (K == 36864)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_64x128x32x3_t0_y1z2_K36864_align8", ([&]
                                                                                               { tir_halfxint3_tensorop_64x128x32x3_t0_y1z2_K36864_align8<<<blocks, threads>>>(
                                                                                                     (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                     (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
    } else if (M >= 256){
            const int warp_size = 32;
            const int warp_row = 2;
            const int warp_col = 2;
            const int raster = 0;
            const int BM = 128;
            const int BN = 256;
            const int BK = 32;
            dim3 blocks(
                (N + BN - 1) / BN, (M + BM - 1) / BM, 1);
            dim3 threads(warp_size, warp_row, warp_col);
            if (K == 9216)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_128x256x32x1_t0_y2z2_K9216_align8", ([&]
                                                                                               { tir_halfxint3_tensorop_128x256x32x1_t0_y2z2_K9216_align8<<<blocks, threads>>>(
                                                                                                     (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                     (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
            else if (K == 36864)
                AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                    mat_A.type(), "tir_halfxint3_tensorop_128x256x32x1_t0_y2z2_K36864_align8", ([&]
                                                                                                { tir_halfxint3_tensorop_128x256x32x1_t0_y2z2_K36864_align8<<<blocks, threads>>>(
                                                                                                      (half *)mat_A.data_ptr(), (signed char *)mat_B.data_ptr(),
                                                                                                      (half *)scales.data_ptr(), (half *)zeros.data_ptr(), (half *)mat_C.data_ptr()); }));
    }
}